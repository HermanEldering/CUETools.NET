/**
 * CUETools.FlaCuda: FLAC audio encoder using CUDA
 * Copyright (c) 2009 Gregory S. Chudov
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#ifndef _FLACUDA_KERNEL_H_
#define _FLACUDA_KERNEL_H_

typedef struct
{
    int samplesOffs;
    int windowOffs;
    int residualOffs;
    int blocksize;
    int reserved[12];
} computeAutocorTaskStruct;

typedef enum
{
    Constant = 0,
    Verbatim = 1,
    Fixed = 8,
    LPC = 32
} SubframeType;

typedef struct
{
    int residualOrder; // <= 32
    int samplesOffs;
    int shift;
    int cbits;
    int size;
    int type;
    int obits;
    int blocksize;
    int best_index;
    int channel;
    int residualOffs;
    int wbits;
    int reserved[4];
    int coefs[32];
} encodeResidualTaskStruct;

extern "C" __global__ void cudaStereoDecorr(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
	samples[2 * offset + pos] = (s.x + s.y) >> 1;
	samples[3 * offset + pos] = s.x - s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr2(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr(
    int *samples,
    short *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
	samples[blockIdx.y * offset + pos] = src[pos * gridDim.y + blockIdx.y];
}

extern "C" __global__ void cudaFindWastedBits(
    encodeResidualTaskStruct *tasks,
    int *samples,
    int tasksPerChannel,
    int blocksize
)
{
    __shared__ struct {
	volatile int wbits[256];
	encodeResidualTaskStruct task;
    } shared;

    if (threadIdx.x < 16)
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.x * tasksPerChannel]))[threadIdx.x];
    shared.wbits[threadIdx.x] = 0;
    __syncthreads();

    for (int pos = 0; pos < blocksize; pos += blockDim.x)
	shared.wbits[threadIdx.x] |= pos + threadIdx.x < blocksize ? samples[shared.task.samplesOffs + pos + threadIdx.x] : 0;
    __syncthreads();

    if (threadIdx.x < 128) shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 128]; __syncthreads();
    if (threadIdx.x < 64) shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 64]; __syncthreads();
    if (threadIdx.x < 32) shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 32]; __syncthreads();
    shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 16];
    shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 8];
    shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 4];
    shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 2];
    shared.wbits[threadIdx.x] |= shared.wbits[threadIdx.x + 1];
    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].wbits = max(0,__ffs(shared.wbits[0]) - 1);
}

extern "C" __global__ void cudaComputeAutocor(
    float *output,
    const int *samples,
    const float *window,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int frameSize,
    int partSize // should be <= 2*blockDim - max_order
)
{
    __shared__ struct {
	float data[512];
	volatile float product[256];
	computeAutocorTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y * 32);
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();

    // fetch samples
    {
	const int pos = blockIdx.x * partSize;
	const int dataLen = min(frameSize - pos, partSize + max_order);

	shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] * window[shared.task.windowOffs + pos + tid]: 0.0f;
	shared.data[tid + 256] = tid + 256 < dataLen ? samples[shared.task.samplesOffs + pos + tid + 256] * window[shared.task.windowOffs + pos + tid + 256]: 0.0f;
    }
    __syncthreads();

    for (int lag = threadIdx.y; lag <= max_order; lag += 8)
    {
        const int productLen = min(frameSize - blockIdx.x * partSize - lag, partSize);
	shared.product[tid] = 0.0;
	for (int ptr = threadIdx.x; ptr < productLen + threadIdx.x; ptr += 128)
	    shared.product[tid] += ((ptr < productLen) * shared.data[ptr] * shared.data[ptr + lag]
				 + (ptr + 32 < productLen) * shared.data[ptr + 32] * shared.data[ptr + 32 + lag])
				 + ((ptr + 64 < productLen) * shared.data[ptr + 64] * shared.data[ptr + 64 + lag]
				 + (ptr + 96 < productLen) * shared.data[ptr + 96] * shared.data[ptr + 96 + lag]);
	// product sum: reduction in shared mem
	//shared.product[tid] += shared.product[tid + 16];
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 16]) + (shared.product[tid + 8] + shared.product[tid + 24]);
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 4]) + (shared.product[tid + 2] + shared.product[tid + 6]);
	// return results
	if (threadIdx.x == 0)
	    output[(blockIdx.x + blockIdx.y * gridDim.x) * (max_order + 1) + lag] = shared.product[tid] + shared.product[tid + 1];
    }
}

extern "C" __global__ void cudaComputeLPC(
    encodeResidualTaskStruct *output,
    float*autoc,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int partCount // should be <= blockDim?
)
{
    __shared__ struct {
	computeAutocorTaskStruct task;
	volatile float ldr[32];
	volatile int   bits[32];
	volatile float autoc[33];
	volatile float gen0[32];
	volatile float gen1[32];
	volatile float parts[128];
	//volatile float reff[32];
	//int   cbits;
    } shared;
    const int tid = threadIdx.x;
    
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    
    // add up parts
    for (int order = 0; order <= max_order; order++)
    {
	shared.parts[tid] = tid < partCount ? autoc[(blockIdx.y * partCount + tid) * (max_order + 1) + order] : 0;
	__syncthreads();
	if (tid < 64 && blockDim.x > 64) shared.parts[tid] += shared.parts[tid + 64];
	__syncthreads();
	if (tid < 32) 
	{
	    if (blockDim.x > 32) shared.parts[tid] += shared.parts[tid + 32];
	    shared.parts[tid] += shared.parts[tid + 16];
	    shared.parts[tid] += shared.parts[tid + 8];
	    shared.parts[tid] += shared.parts[tid + 4];
	    shared.parts[tid] += shared.parts[tid + 2];
	    shared.parts[tid] += shared.parts[tid + 1];
	    if (tid == 0)
		shared.autoc[order] = shared.parts[0];
	}
    }
   
    if (tid < 32)
    {
	shared.gen0[tid] = shared.autoc[tid+1];
	shared.gen1[tid] = shared.autoc[tid+1];
	shared.ldr[tid] = 0.0f;

	float error = shared.autoc[0];
	for (int order = 0; order < max_order; order++)
	{
	    // Schur recursion
	    float reff = -shared.gen1[0] / error;
	    //if (tid == 0) shared.reff[order] = reff;
	    error += __fmul_rz(shared.gen1[0], reff);
	    if (tid < max_order - 1 - order)
	    {
		float g1 = shared.gen1[tid + 1] + __fmul_rz(reff, shared.gen0[tid]);
		float g0 = __fmul_rz(shared.gen1[tid + 1], reff) + shared.gen0[tid];
		shared.gen1[tid] = g1;
		shared.gen0[tid] = g0;
	    }

	    // Levinson-Durbin recursion
	    shared.ldr[tid] += (tid < order) * __fmul_rz(reff, shared.ldr[order - 1 - tid]) + (tid  == order) * reff;

	    // Quantization
	    int precision = 13 - (order > 8);
	    int taskNo = shared.task.residualOffs + order;
	    shared.bits[tid] = __mul24((33 - __clz(__float2int_rn(fabs(shared.ldr[tid]) * (1 << 15))) - precision), tid <= order);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int sh = max(0,min(15, 15 - shared.bits[0]));
	    
	    // reverse coefs
	    int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(-shared.ldr[order - tid] * (1 << sh))));
	    if (tid <= order)
		output[taskNo].coefs[tid] = coef;
	    if (tid == 0)
		output[taskNo].shift = sh;
	    shared.bits[tid] = 33 - max(__clz(coef),__clz(-1 ^ coef));
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int cbits = shared.bits[0];
	    if (tid == 0)
		output[taskNo].cbits = cbits;
	}
    }
}

#define SUM32(buf,tid)      buf[tid] += buf[tid + 16]; buf[tid] += buf[tid + 8]; buf[tid] += buf[tid + 4]; buf[tid] += buf[tid + 2]; buf[tid] += buf[tid + 1];
#define SUM64(buf,tid)      if (tid < 32) buf[tid] += buf[tid + 32]; __syncthreads(); if (tid < 32) SUM32(buf,tid)
#define SUM128(buf,tid)     if (tid < 64) buf[tid] += buf[tid + 64]; __syncthreads(); SUM64(buf,tid)
#define SUM256(buf,tid)     if (tid < 128) buf[tid] += buf[tid + 128]; __syncthreads(); SUM128(buf,tid)
#define SUM512(buf,tid)     if (tid < 256) buf[tid] += buf[tid + 256]; __syncthreads(); SUM256(buf,tid)

#define FSQR(s) ((s)*(s))

extern "C" __global__ void cudaComputeLPCLattice(
    encodeResidualTaskStruct *tasks,
    const int taskCount, // tasks per block
    const int *samples,
    const int frameSize, // <= 512
    const int max_order // should be <= 32
)
{
    __shared__ struct {
	encodeResidualTaskStruct task;
	volatile float F[512];
	volatile float B[512];
	volatile float tmp[256];
	volatile float arp[32];
	volatile float rc[32];
	volatile int   bits[32];
	volatile float PE[33];
	volatile float DEN, reff;
    } shared;

    // fetch task data
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];    
    __syncthreads();

    // F = samples; B = samples
    shared.F[threadIdx.x] = threadIdx.x < frameSize ? samples[shared.task.samplesOffs + threadIdx.x] >> shared.task.wbits : 0.0f;
    shared.F[threadIdx.x + 256] = threadIdx.x + 256 < frameSize ? samples[shared.task.samplesOffs + threadIdx.x + 256] >> shared.task.wbits : 0.0f;
    shared.B[threadIdx.x] = shared.F[threadIdx.x];
    shared.B[threadIdx.x + 256] = shared.F[threadIdx.x + 256];
    __syncthreads();

    // DEN = F*F'
    shared.tmp[threadIdx.x] = FSQR(shared.F[threadIdx.x]) + FSQR(shared.F[threadIdx.x + 256]);
    __syncthreads();
    SUM256(shared.tmp,threadIdx.x);
    if (threadIdx.x == 0)
    {
	shared.DEN = shared.tmp[0];
	shared.PE[0] = shared.tmp[0] / frameSize;
    }
    __syncthreads();

    for (int order = 1; order <= max_order; order++)
    {
	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
	float f1 = (threadIdx.x + order < frameSize) * shared.F[order + threadIdx.x];
	float f2 = (threadIdx.x + 256 + order < frameSize) * shared.F[order + threadIdx.x + 256];
	shared.tmp[threadIdx.x] = f1 * shared.B[threadIdx.x] + f2 * shared.B[threadIdx.x + 256]; 
	__syncthreads(); 
	SUM256(shared.tmp, threadIdx.x);
	if (threadIdx.x == 0)
	    shared.reff = shared.tmp[0] / shared.DEN;
	__syncthreads();

	// arp(order) = rc(order) = reff
	if (threadIdx.x == 0)
	    shared.arp[order - 1] = shared.rc[order - 1] = shared.reff;

	// Levinson-Durbin recursion
	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
	if (threadIdx.x < 32)
	    shared.arp[threadIdx.x] -= (threadIdx.x < order - 1) * __fmul_rz(shared.reff, shared.arp[order - 2 - threadIdx.x]);

	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
	// F(order+1:frameSize) = F1
	if (threadIdx.x < frameSize - order)
	{
	    shared.F[order + threadIdx.x] -= shared.reff * shared.B[threadIdx.x];
	    shared.B[threadIdx.x] -= shared.reff * f1;
	}
	if (threadIdx.x + 256 < frameSize - order)
	{
	    shared.F[order + threadIdx.x + 256] -= shared.reff * shared.B[threadIdx.x + 256];
	    shared.B[threadIdx.x + 256] -= shared.reff * f2;
	}
	__syncthreads();

	// DEN = F(order+1:frameSize) * F(order+1:frameSize)' + B(1:frameSize-order) * B(1:frameSize-order)' (BURG)
	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * (FSQR(shared.F[threadIdx.x + order]) + FSQR(shared.B[threadIdx.x]))
	    + (threadIdx.x + 256 < frameSize - order) * (FSQR(shared.F[threadIdx.x + 256 + order]) + FSQR(shared.B[threadIdx.x + 256]));
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x);
	if (threadIdx.x == 0)
	{
	    shared.DEN = shared.tmp[0] / 2;
	    shared.PE[order] = shared.tmp[0] / 2 / (frameSize - order);
	}
	__syncthreads();

	// Quantization
	if (threadIdx.x < 32)
	{
	    int precision = 10 - (order > 8) - min(2, shared.task.wbits);
	    int taskNo = taskCount * blockIdx.y + order - 1;
	    shared.bits[threadIdx.x] = __mul24((33 - __clz(__float2int_rn(fabs(shared.arp[threadIdx.x]) * (1 << 15))) - precision), threadIdx.x < order);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
	    int sh = max(0,min(15, 15 - shared.bits[0]));
            
	    // reverse coefs
	    int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(shared.arp[order - 1 - threadIdx.x] * (1 << sh))));
	    if (threadIdx.x < order)
		tasks[taskNo].coefs[threadIdx.x] = coef;
	    if (threadIdx.x == 0)
		tasks[taskNo].shift = sh;
	    shared.bits[threadIdx.x] = 33 - max(__clz(coef),__clz(-1 ^ coef));
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
	    shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
	    int cbits = shared.bits[0];
	    if (threadIdx.x == 0)
		tasks[taskNo].cbits = cbits;
	}
    }
}

extern "C" __global__ void cudaComputeLPCLattice512(
    encodeResidualTaskStruct *tasks,
    const int taskCount, // tasks per block
    const int *samples,
    const int frameSize, // <= 512
    const int max_order // should be <= 32
)
{
    __shared__ struct {
	encodeResidualTaskStruct task;
	float F[512];
	float B[512];
	float lpc[32][32];
	volatile float tmp[512];
	volatile float arp[32];
	volatile float rc[32];
	volatile int   bits[512];
	volatile float f, b;
    } shared;

    // fetch task data
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];    
    __syncthreads();

    // F = samples; B = samples
    shared.F[threadIdx.x] = threadIdx.x < frameSize ? samples[shared.task.samplesOffs + threadIdx.x] >> shared.task.wbits : 0.0f;
    shared.B[threadIdx.x] = shared.F[threadIdx.x];
    __syncthreads();

    // DEN = F*F'
    shared.tmp[threadIdx.x] = FSQR(shared.F[threadIdx.x]);
    __syncthreads();
    SUM512(shared.tmp,threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0)
	shared.f = shared.b = shared.tmp[0];
 //   if (threadIdx.x == 0)
	//shared.PE[0] = DEN / frameSize;
    __syncthreads();

    for (int order = 1; order <= max_order; order++)
    {
	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
	shared.tmp[threadIdx.x] = (threadIdx.x + order < frameSize) * shared.F[threadIdx.x + order] * shared.B[threadIdx.x];
	__syncthreads(); 
	SUM512(shared.tmp, threadIdx.x);
	__syncthreads();
	
	//float reff = shared.tmp[0] * rsqrtf(shared.b * shared.f); // Geometric lattice
	float reff = shared.tmp[0] * 2 / (shared.b + shared.f); // Burg method
	__syncthreads();

	// Levinson-Durbin recursion
	// arp(order) = rc(order) = reff
	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
	if (threadIdx.x == 32)
	    shared.arp[order - 1] = shared.rc[order - 1] = reff;
	if (threadIdx.x < 32)
	    shared.arp[threadIdx.x] -= (threadIdx.x < order - 1) * __fmul_rz(reff, shared.arp[order - 2 - threadIdx.x]);

	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
	// F(order+1:frameSize) = F1
	if (threadIdx.x < frameSize - order)
	{
	    float f;// = shared.F[threadIdx.x + order];
	    shared.F[threadIdx.x + order] = (f = shared.F[threadIdx.x + order]) - reff * shared.B[threadIdx.x];
	    shared.B[threadIdx.x] -= reff * f;
	}
	__syncthreads();

	// f = F(order+1:frameSize) * F(order+1:frameSize)'
	// b = B(1:frameSize-order) * B(1:frameSize-order)'
	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.F[threadIdx.x + order]);
	__syncthreads();
	SUM512(shared.tmp, threadIdx.x);
	__syncthreads();
	if (threadIdx.x == 0)
	    shared.f = shared.tmp[0];
	__syncthreads();

	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.B[threadIdx.x]);
	__syncthreads();
	SUM512(shared.tmp, threadIdx.x);
	__syncthreads();
	if (threadIdx.x == 0)
	    shared.b = shared.tmp[0];
	__syncthreads();

	if (threadIdx.x < 32)
	    shared.lpc[order - 1][threadIdx.x] = shared.arp[threadIdx.x];

	//if (threadIdx.x == 0)
	//    shared.PE[order] = (shared.b + shared.f) / 2 / (frameSize - order);
	__syncthreads();
    }
    for (int order = 1 + (threadIdx.x >> 5); order <= max_order; order += 16)
    {
	// Quantization
	int cn = threadIdx.x & 31;
	int precision = 10 - (order > 8) - min(2, shared.task.wbits);
	int taskNo = taskCount * blockIdx.y + order - 1;
	shared.bits[threadIdx.x] = __mul24((33 - __clz(__float2int_rn(fabs(shared.lpc[order - 1][cn]) * (1 << 15))) - precision), cn < order);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
	int sh = max(0,min(15, 15 - shared.bits[threadIdx.x - cn]));
            
	// reverse coefs
	int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(shared.lpc[order - 1][order - 1 - cn] * (1 << sh))));
	if (cn < order)
	    tasks[taskNo].coefs[cn] = coef;
	if (cn == 0)
	    tasks[taskNo].shift = sh;
	shared.bits[threadIdx.x] = 33 - max(__clz(coef),__clz(-1 ^ coef));
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
	int cbits = shared.bits[threadIdx.x - cn];
	if (cn == 0)
	    tasks[taskNo].cbits = cbits;
    }
}

// blockDim.x == 32
// blockDim.y == 8
extern "C" __global__ void cudaEstimateResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks,
    int max_order,
    int frameSize,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	int data[32*9];
	volatile int residual[32*8];
	encodeResidualTaskStruct task[8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (threadIdx.x < 16)
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[blockIdx.y * blockDim.y + threadIdx.y]))[threadIdx.x];
    __syncthreads();
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(frameSize - pos, partSize + max_order);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task[0].samplesOffs + pos + tid] >> shared.task[0].wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task[0].samplesOffs + pos + tid + partSize] >> shared.task[0].wbits : 0;
    const int residualLen = max(0,min(frameSize - pos - shared.task[threadIdx.y].residualOrder, partSize));

    __syncthreads();

    shared.residual[tid] = 0;
    shared.task[threadIdx.y].coefs[threadIdx.x] = threadIdx.x < max_order ? tasks[blockIdx.y * blockDim.y + threadIdx.y].coefs[threadIdx.x] : 0;

    for (int i = blockDim.y * (shared.task[threadIdx.y].type == Verbatim); i < blockDim.y; i++) // += 32
    {
	int ptr = threadIdx.x + (i<<5);
	// compute residual
	int sum = 0;
	int c = 0;
	for (c = 0; c < shared.task[threadIdx.y].residualOrder; c++)
	    sum += __mul24(shared.data[ptr + c], shared.task[threadIdx.y].coefs[c]);
	sum = shared.data[ptr + c] - (sum >> shared.task[threadIdx.y].shift);
	shared.residual[tid] += __mul24(ptr < residualLen, min(0x7fffff,(sum << 1) ^ (sum >> 31)));
    }

    // enable this line when using blockDim.x == 64
    //__syncthreads(); if (threadIdx.x < 32) shared.residual[tid] += shared.residual[tid + 32]; __syncthreads();
    shared.residual[tid] += shared.residual[tid + 16];
    shared.residual[tid] += shared.residual[tid + 8];
    shared.residual[tid] += shared.residual[tid + 4];
    shared.residual[tid] += shared.residual[tid + 2];
    shared.residual[tid] += shared.residual[tid + 1];

    // rice parameter search
    shared.residual[tid] = (shared.task[threadIdx.y].type != Constant || shared.residual[threadIdx.y * blockDim.x] != 0) *
	(__mul24(threadIdx.x >= 15, 0x7fffff) + residualLen * (threadIdx.x + 1) + ((shared.residual[threadIdx.y * blockDim.x] - (residualLen >> 1)) >> threadIdx.x));
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 8]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 4]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 2]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 1]);
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid];
}

#define BEST_INDEX(a,b) ((a) + ((b) - (a)) * (shared.length[b] < shared.length[a]))

extern "C" __global__ void cudaChooseBestMethod(
    encodeResidualTaskStruct *tasks,
    int *residual,
    int partCount, // <= blockDim.y (256)
    int taskCount
    )
{
    __shared__ struct {
	volatile int index[128];
	volatile int partLen[512];
	int length[256];
	volatile encodeResidualTaskStruct task[16];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    
    if (tid < 256) shared.length[tid] = 0x7fffffff;
    for (int task = 0; task < taskCount; task += blockDim.y)
	if (task + threadIdx.y < taskCount)
	{
	    // fetch task data
	    ((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(tasks + task + threadIdx.y + taskCount * blockIdx.y))[threadIdx.x];

	    int sum = 0;
	    for (int pos = 0; pos < partCount; pos += blockDim.x)
		sum += (pos + threadIdx.x < partCount ? residual[pos + threadIdx.x + 64 * (task + threadIdx.y + taskCount * blockIdx.y)] : 0);
	    shared.partLen[tid] = sum;

	    // length sum: reduction in shared mem
	    shared.partLen[tid] += shared.partLen[tid + 16];
	    shared.partLen[tid] += shared.partLen[tid + 8];
	    shared.partLen[tid] += shared.partLen[tid + 4];
	    shared.partLen[tid] += shared.partLen[tid + 2];
	    shared.partLen[tid] += shared.partLen[tid + 1];
	    // return sum
	    if (threadIdx.x == 0)
	    {
		int obits = shared.task[threadIdx.y].obits - shared.task[threadIdx.y].wbits;
		shared.length[task + threadIdx.y] =
		    min(obits * shared.task[threadIdx.y].blocksize,
			shared.task[threadIdx.y].type == Fixed ? shared.task[threadIdx.y].residualOrder * obits + 6 + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == LPC ? shared.task[threadIdx.y].residualOrder * obits + 4 + 5 + shared.task[threadIdx.y].residualOrder * shared.task[threadIdx.y].cbits + 6 + (4 * partCount/2)/* << porder */ + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == Constant ? obits * (1 + shared.task[threadIdx.y].blocksize * (shared.partLen[threadIdx.y * 32] != 0)) : 
			obits * shared.task[threadIdx.y].blocksize);
	    }
	}
    //shared.index[threadIdx.x] = threadIdx.x;
    //shared.length[threadIdx.x] = (threadIdx.x < taskCount) ? tasks[threadIdx.x + taskCount * blockIdx.y].size : 0x7fffffff;

    __syncthreads();

    //if (tid < 128) shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 128]); __syncthreads();
    if (tid < 128) shared.index[tid] = BEST_INDEX(tid, tid + 128); __syncthreads();
    if (tid < 64) shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 64]); __syncthreads();
    if (tid < 32) 
    {
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 32]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 16]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 8]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 4]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 2]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 1]);
    }
    __syncthreads();
 //   if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	//((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y + shared.index[0]))[threadIdx.x];
    if (tid == 0)
	tasks[taskCount * blockIdx.y].best_index = taskCount * blockIdx.y + shared.index[0];
    if (tid < taskCount)
	tasks[tid + taskCount * blockIdx.y].size = shared.length[tid];
}

extern "C" __global__ void cudaCopyBestMethod(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index;
    } shared;
    if (threadIdx.x == 0)
	shared.best_index = tasks[count * blockIdx.y].best_index;
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.best_index))[threadIdx.x];
}

extern "C" __global__ void cudaCopyBestMethodStereo(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index[4];
	int best_size[4];
	int lr_index[2];
    } shared;
    if (threadIdx.x < 4)
	shared.best_index[threadIdx.x] = tasks[count * (blockIdx.y * 4 + threadIdx.x)].best_index;
    if (threadIdx.x < 4)
	shared.best_size[threadIdx.x] = tasks[shared.best_index[threadIdx.x]].size;
    __syncthreads();
    if (threadIdx.x == 0)
    {
	int bitsBest = 0x7fffffff;
	if (bitsBest > shared.best_size[2] + shared.best_size[3]) // MidSide
	{
	    bitsBest = shared.best_size[2] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[2];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[3] + shared.best_size[1]) // RightSide
	{
	    bitsBest = shared.best_size[3] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[3];
	    shared.lr_index[1] = shared.best_index[1];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[3]) // LeftSide
	{
	    bitsBest = shared.best_size[0] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[1]) // LeftRight
	{
	    bitsBest = shared.best_size[0] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[1];
	}
    }
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.lr_index[0]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y].residualOffs = tasks[shared.best_index[0]].residualOffs;
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y + 1))[threadIdx.x] = ((int*)(tasks + shared.lr_index[1]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y + 1].residualOffs = tasks[shared.best_index[1]].residualOffs;
}

extern "C" __global__ void cudaEncodeResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks
    )
{
    __shared__ struct {
	int data[256 + 32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x;
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    __syncthreads();
    const int partSize = blockDim.x;
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(shared.task.blocksize - pos, partSize + shared.task.residualOrder);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] >> shared.task.wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task.samplesOffs + pos + tid + partSize] >> shared.task.wbits : 0;
    const int residualLen = max(0,min(shared.task.blocksize - pos - shared.task.residualOrder, partSize));

    __syncthreads();
    
    // compute residual
    int sum = 0;
    for (int c = 0; c < shared.task.residualOrder; c++)
	sum += __mul24(shared.data[tid + c], shared.task.coefs[c]);
    if (tid < residualLen)
	output[shared.task.residualOffs + pos + tid] = shared.data[tid + shared.task.residualOrder] - (sum >> shared.task.shift);
}
#endif
