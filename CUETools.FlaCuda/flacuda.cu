/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#ifndef _FLACUDA_KERNEL_H_
#define _FLACUDA_KERNEL_H_

extern "C" __global__ void cudaComputeAutocor(
    float *output,
    const int *samples,
    const float *window,
    int frameSize,
    int frameOffset,
    int blocks)
{
    __shared__ struct {
	float data[512];
	float matrix[512];
    } shared;
    const int iWin = blockIdx.y >> 2;
    const int iCh = blockIdx.y & 3;
    const int smpBase = iCh * frameOffset;
    const int winBase = iWin * 2 * frameOffset;
    const int pos = blockIdx.x * blocks;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // fetch blockDim.x*blockDim.y samples
    shared.data[tid] = pos + tid < frameSize ? samples[smpBase + pos + tid] * window[winBase + pos + tid] : 0.0f;
    __syncthreads();

    float s = 0.0f;    
    for (int i = 0; i < blocks; i += blockDim.y) 
	s += shared.data[i + threadIdx.y] * shared.data[i + threadIdx.y + threadIdx.x];
    shared.matrix[tid] = s;
    __syncthreads();

    // reduction in shared mem
    for(unsigned int s=blockDim.y/2; s>1; s>>=1)
    {
	if (threadIdx.y < s)
	    shared.matrix[tid] += shared.matrix[tid + s * blockDim.x];
	__syncthreads();
    }

    // return results
    if (threadIdx.y == 0)
	output[(blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x] = shared.matrix[threadIdx.x] + shared.matrix[threadIdx.x + blockDim.x];
}

typedef struct
{
    int residualOrder;
    int shift;
    int coefsOffs;
    int samplesOffs;
} encodeResidualTaskStruct;

extern "C" __global__ void cudaEncodeResidual(
    int*output,
    int*samples,
    int*allcoefs,
    encodeResidualTaskStruct *tasks,
    int frameSize,
    int partSize
    )
{
    __shared__ struct {
	int data[256];
	int residual[256];
	int coefs[32];
	int rice[32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x;
    // fetch task data
    if (tid < sizeof(encodeResidualTaskStruct) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();
    const int pos = blockIdx.x * partSize;
    const int residualOrder = shared.task.residualOrder;
    const int dataLen = min(frameSize - pos, partSize + residualOrder + 1);
    const int residualLen = dataLen - residualOrder - 1;

    // fetch coeffs, inverting their order
    if (tid <= residualOrder) shared.coefs[residualOrder - tid] = allcoefs[shared.task.coefsOffs + tid];
    // fetch samples
    shared.data[tid] = (tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] : 0);
    
    // compute residual
    __syncthreads();
    long sum = 0;
    for (int c = 0; c <= residualOrder; c++)
	sum += __mul24(shared.data[tid + c], shared.coefs[c]);
    int res = shared.data[tid + residualOrder + 1] - (sum >> shared.task.shift);
    shared.residual[tid] = __mul24(tid < residualLen, (2 * res) ^ (res >> 31));
    
    __syncthreads();
    // residual sum: reduction in shared mem
    if (tid < 128) shared.residual[tid] += shared.residual[tid + 128]; __syncthreads();
    if (tid < 64) shared.residual[tid] += shared.residual[tid + 64]; __syncthreads();
    if (tid < 32) shared.residual[tid] += shared.residual[tid + 32]; __syncthreads();
    shared.residual[tid] += shared.residual[tid + 16];
    shared.residual[tid] += shared.residual[tid + 8];
    shared.residual[tid] += shared.residual[tid + 4];
    shared.residual[tid] += shared.residual[tid + 2];
    shared.residual[tid] += shared.residual[tid + 1];
    __syncthreads();

    if (tid < 32)
    {
	// rice parameter search
	shared.rice[tid] = __mul24(tid >= 15, 0x7fffff) + residualLen * (tid + 1) + ((shared.residual[0] - (residualLen >> 1)) >> tid);
	shared.rice[tid] = min(shared.rice[tid], shared.rice[tid + 8]);
	shared.rice[tid] = min(shared.rice[tid], shared.rice[tid + 4]);
	shared.rice[tid] = min(shared.rice[tid], shared.rice[tid + 2]);
	shared.rice[tid] = min(shared.rice[tid], shared.rice[tid + 1]);
    }
    if (tid == 0)
	output[blockIdx.x + blockIdx.y * gridDim.x] = shared.rice[0];
}

#if 0
extern "C" __global__ void cudaComputeAutocor3int(const int * samples, const float * window, int* output, int frameSize, int frameOffset, int channels)
{
    extern __shared__ short shared[];
    int *ishared = (int*)shared;
    const int lag = blockIdx.x;
    
    // fetch signal, multiply by window and split high bits/low bits
    const int iWin = blockIdx.y >> 2; // blockIdx.y/channels;
    const int iCh = (blockIdx.y  - iWin * channels);
    const int smpBase = iCh * frameOffset;
    const int winBase = iWin * 2 * frameOffset;

    for(int i = threadIdx.x; i < frameSize; i += blockDim.x)
    {
	float val = samples[smpBase + i] * window[winBase + i];
	int ival = __float2int_rz(fabs(val));
	int sg =  (1 - 2 *signbit(val));
	//int ival = (int) val;
	//int sg =  ival < 0 ? -1 : 1;
	ival = ival < 0 ? -ival : ival;
	shared[i*2] = __mul24(sg, (ival >> 9));
	shared[i*2+1] = __mul24(sg, (ival & ((1 << 9) - 1)));
    }
    __syncthreads();

    // correlation
    int sum1 = 0;
    int sum2 = 0;
    int sum3 = 0;
    for (int i = threadIdx.x; i < frameSize - lag; i += blockDim.x)
    {
	sum1 += __mul24(shared[2*i], shared[2*(i+lag)]);
	sum2 += __mul24(shared[2*i+1], shared[2*(i+lag)]);
	sum2 += __mul24(shared[2*i], shared[2*(i+lag)+1]);
	sum3 += __mul24(shared[2*i+1], shared[2*(i+lag)+1]);
    }    
    __syncthreads();
    ishared[threadIdx.x] = sum1;
    ishared[threadIdx.x + blockDim.x] = sum2;
    ishared[threadIdx.x + blockDim.x * 2] = sum3;
    __syncthreads();

    // reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (threadIdx.x < s)
        {
            ishared[threadIdx.x] += ishared[threadIdx.x + s];
	    ishared[threadIdx.x + blockDim.x] += ishared[threadIdx.x + s + blockDim.x];
	    ishared[threadIdx.x + blockDim.x * 2] += ishared[threadIdx.x + s + blockDim.x * 2];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
	output[(blockIdx.x + blockIdx.y * gridDim.x) * 3] = ishared[threadIdx.x];
	output[(blockIdx.x + blockIdx.y * gridDim.x) * 3 + 1] = ishared[threadIdx.x + blockDim.x];
	output[(blockIdx.x + blockIdx.y * gridDim.x) * 3 + 2] = ishared[threadIdx.x + blockDim.x * 2];
    } 
}

__device__ float Bartlett(int i, int blockSize)
{
    float n = fminf(i, blockSize - i);
    float k = 2.0f / blockSize * (blockSize / 2.0f - n);
    k = 1.0f - k * k;
    return k*k;
}

extern "C" __global__ void cudaComputeAutocorPart(const int * samples, const float * window, float* output, int frameSize, int frameOffset, int iCh, int iWin)
{
    extern __shared__ float fshared[];
    // fetch signal, multiply by window
    //const int iWin = blockIdx.y;
    //const int iCh = blockIdx.x;
    const int smpBase = iCh * frameOffset;
    const int winBase = iWin * 2 * frameOffset;
    float * matrix = fshared + 129;

    // initialize results matrix
    matrix[threadIdx.x + threadIdx.y * (blockDim.x + 1)] = 0.0f;

    // prefetch blockDim.x + blockDim.y samples
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < blockDim.x + blockDim.y)
    {
	if (blockIdx.x * blockDim.y + tid < frameSize)
	    fshared[tid] = samples[smpBase + blockIdx.x * blockDim.y + tid] * window[winBase + blockIdx.x * blockDim.y + tid];
	else
	    fshared[tid] = 0.0f;
    }

    __syncthreads();

    matrix[threadIdx.x + threadIdx.y * (1 + blockDim.x)] += fshared[threadIdx.y] * fshared[threadIdx.y + threadIdx.x];

    __syncthreads();

    // reduction in shared mem
    for(unsigned int s=blockDim.y/2; s>0; s>>=1)
    {
	if (threadIdx.y < s)
	    matrix[threadIdx.x + threadIdx.y * (1 + blockDim.x)] += matrix[threadIdx.x + (s + threadIdx.y) * (1 + blockDim.x)];
	__syncthreads();
    }

    // return results
    if (threadIdx.y == 0)
	output[blockIdx.x * blockDim.x + threadIdx.x] = matrix[threadIdx.x];
}

extern "C" __global__ void cudaComputeAutocor2(const int * samples, const float * window, float* output, int frameSize, int frameOffset)
{
    extern __shared__ float fshared[];   
    // fetch signal, multiply by window
    const int iWin = blockIdx.y;
    const int iCh = blockIdx.x;
    const int smpBase = iCh * frameOffset;
    const int winBase = iWin * 2 * frameOffset;

    for(int i = threadIdx.x + threadIdx.y * blockDim.x; i < frameSize; i += blockDim.x * blockDim.y)
	fshared[i] = samples[smpBase + i] * window[winBase + i];

    __syncthreads();

    const int lag = threadIdx.y;

    // correlation
    float sum = 0.0f;
    for (int i = threadIdx.x; i < frameSize - lag; i += blockDim.x)
	sum += fshared[i] * fshared[i+lag];
    __syncthreads();

    fshared[threadIdx.x + threadIdx.y * blockDim.x] = sum;

    __syncthreads();

    // reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (threadIdx.x < s)
            fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x + s + threadIdx.y * blockDim.x];
        __syncthreads();
    }

 //   if (threadIdx.x < 32)
 //   {
	//if (blockDim.x >= 64) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x + 32 + threadIdx.y * blockDim.x];
	//if (blockDim.x >= 32) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x + 16 + threadIdx.y * blockDim.x];
	//if (blockDim.x >= 16) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x +  8 + threadIdx.y * blockDim.x];
	//if (blockDim.x >=  8) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x +  4 + threadIdx.y * blockDim.x];
	//if (blockDim.x >=  4) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x +  2 + threadIdx.y * blockDim.x];
	//if (blockDim.x >=  2) fshared[threadIdx.x + threadIdx.y * blockDim.x] += fshared[threadIdx.x +  1 + threadIdx.y * blockDim.x];
 //   }

    if (threadIdx.x == 0) {
	output[(blockIdx.x + blockIdx.y * gridDim.x) * blockDim.y + threadIdx.y] 
	= fshared[threadIdx.x + threadIdx.y * blockDim.x];
    } 
}

extern "C" __global__ void cudaComputeAutocorOld(const int * samples, const float * window, float* output, int frameSize, int frameOffset, int channels)
{
    extern __shared__ float fshared[];
    const int lag = blockIdx.x;
    
    // fetch signal, multiply by window
    const int iWin = blockIdx.y >> 2; // blockIdx.y/channels;
    const int iCh = (blockIdx.y  - iWin * channels);
    const int smpBase = iCh * frameOffset;
    const int winBase = iWin * 2 * frameOffset;

    for(int i = threadIdx.x; i < frameSize; i += blockDim.x)
	fshared[i] = samples[smpBase + i] * window[winBase + i];

    __syncthreads();

    // correlation
    float sum = 0.0f;
    for (int i = threadIdx.x; i < frameSize - lag; i += blockDim.x)
	sum += fshared[i] * fshared[i+lag];
    __syncthreads();

    fshared[threadIdx.x] = sum;

    __syncthreads();

    // reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (threadIdx.x < s)
            fshared[threadIdx.x] += fshared[threadIdx.x + s];
        __syncthreads();
    }

    if (threadIdx.x == 0) {
	output[blockIdx.x + blockIdx.y * gridDim.x] = fshared[threadIdx.x];
    } 
}
#endif

#endif // _FLACUDA_KERNEL_H_
