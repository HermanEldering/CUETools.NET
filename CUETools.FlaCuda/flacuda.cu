/**
 * CUETools.FlaCuda: FLAC audio encoder using CUDA
 * Copyright (c) 2009 Gregory S. Chudov
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#ifndef _FLACUDA_KERNEL_H_
#define _FLACUDA_KERNEL_H_

typedef struct
{
    int samplesOffs;
    int windowOffs;
    int residualOffs;
    int blocksize;
    int reserved[12];
} computeAutocorTaskStruct;

typedef enum
{
    Constant = 0,
    Verbatim = 1,
    Fixed = 8,
    LPC = 32
} SubframeType;

typedef struct
{
    int residualOrder; // <= 32
    int samplesOffs;
    int shift;
    int cbits;
    int size;
    int type;
    int obits;
    int blocksize;
    int best_index;
    int channel;
    int residualOffs;
    int wbits;
    int abits;
    int reserved[3];
    int coefs[32];
} encodeResidualTaskStruct;

#define SUM16(buf,tid,op)   buf[tid] op buf[tid + 8]; buf[tid] op buf[tid + 4]; buf[tid] op buf[tid + 2]; buf[tid] op buf[tid + 1];
#define SUM32(buf,tid,op)   buf[tid] op buf[tid + 16]; SUM16(buf,tid,op)
#define SUM64(buf,tid,op)   if (tid < 32) buf[tid] op buf[tid + 32]; __syncthreads(); if (tid < 32) { SUM32(buf,tid,op) }
#define SUM128(buf,tid,op)  if (tid < 64) buf[tid] op buf[tid + 64]; __syncthreads(); SUM64(buf,tid,op)
#define SUM256(buf,tid,op)  if (tid < 128) buf[tid] op buf[tid + 128]; __syncthreads(); SUM128(buf,tid,op)
#define SUM512(buf,tid,op)  if (tid < 256) buf[tid] op buf[tid + 256]; __syncthreads(); SUM256(buf,tid,op)

#define FSQR(s) ((s)*(s))

extern "C" __global__ void cudaStereoDecorr(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
	samples[2 * offset + pos] = (s.x + s.y) >> 1;
	samples[3 * offset + pos] = s.x - s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr2(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr(
    int *samples,
    short *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
	samples[blockIdx.y * offset + pos] = src[pos * gridDim.y + blockIdx.y];
}

extern "C" __global__ void cudaFindWastedBits(
    encodeResidualTaskStruct *tasks,
    int *samples,
    int tasksPerChannel,
    int blocksize
)
{
    __shared__ struct {
	volatile int wbits[256];
	volatile int abits[256];
	encodeResidualTaskStruct task;
    } shared;

    if (threadIdx.x < 16)
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.x * tasksPerChannel]))[threadIdx.x];
    shared.wbits[threadIdx.x] = 0;
    shared.abits[threadIdx.x] = 0;
    __syncthreads();

    for (int pos = 0; pos < blocksize; pos += blockDim.x)
    {
	int smp = pos + threadIdx.x < blocksize ? samples[shared.task.samplesOffs + pos + threadIdx.x] : 0;
	shared.wbits[threadIdx.x] |= smp;
	shared.abits[threadIdx.x] |= smp ^ (smp >> 31);
    }
    __syncthreads();
    SUM256(shared.wbits, threadIdx.x, |=);
    SUM256(shared.abits, threadIdx.x, |=);
    if (threadIdx.x == 0)
	shared.task.wbits = max(0,__ffs(shared.wbits[0]) - 1);
    if (threadIdx.x == 0)
	shared.task.abits = 32 - __clz(shared.abits[0]) - shared.task.wbits;
    __syncthreads();

    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].wbits = shared.task.wbits;
    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].abits = shared.task.abits;
}

extern "C" __global__ void cudaComputeAutocor(
    float *output,
    const int *samples,
    const float *window,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int frameSize,
    int partSize // should be <= 2*blockDim - max_order
)
{
    __shared__ struct {
	float data[512];
	volatile float product[256];
	computeAutocorTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y * 32);
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();

    // fetch samples
    {
	const int pos = blockIdx.x * partSize;
	const int dataLen = min(frameSize - pos, partSize + max_order);

	shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] * window[shared.task.windowOffs + pos + tid]: 0.0f;
	shared.data[tid + 256] = tid + 256 < dataLen ? samples[shared.task.samplesOffs + pos + tid + 256] * window[shared.task.windowOffs + pos + tid + 256]: 0.0f;
    }
    __syncthreads();

    for (int lag = threadIdx.y; lag <= max_order; lag += 8)
    {
        const int productLen = min(frameSize - blockIdx.x * partSize - lag, partSize);
	shared.product[tid] = 0.0;
	for (int ptr = threadIdx.x; ptr < productLen + threadIdx.x; ptr += 128)
	    shared.product[tid] += ((ptr < productLen) * shared.data[ptr] * shared.data[ptr + lag]
				 + (ptr + 32 < productLen) * shared.data[ptr + 32] * shared.data[ptr + 32 + lag])
				 + ((ptr + 64 < productLen) * shared.data[ptr + 64] * shared.data[ptr + 64 + lag]
				 + (ptr + 96 < productLen) * shared.data[ptr + 96] * shared.data[ptr + 96 + lag]);
	// product sum: reduction in shared mem
	//shared.product[tid] += shared.product[tid + 16];
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 16]) + (shared.product[tid + 8] + shared.product[tid + 24]);
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 4]) + (shared.product[tid + 2] + shared.product[tid + 6]);
	// return results
	if (threadIdx.x == 0)
	    output[(blockIdx.x + blockIdx.y * gridDim.x) * (max_order + 1) + lag] = shared.product[tid] + shared.product[tid + 1];
    }
}

extern "C" __global__ void cudaComputeLPC(
    encodeResidualTaskStruct *output,
    float*autoc,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int partCount // should be <= blockDim?
)
{
    __shared__ struct {
	computeAutocorTaskStruct task;
	encodeResidualTaskStruct task2;
	volatile float ldr[32];
	volatile int   bits[32];
	volatile float autoc[33];
	volatile float gen0[32];
	volatile float gen1[32];
	volatile float parts[128];
	//volatile float reff[32];
	//int   cbits;
    } shared;
    const int tid = threadIdx.x;
    
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();
    if (tid < sizeof(shared.task2) / sizeof(int))
	((int*)&shared.task2)[tid] = ((int*)(output + shared.task.residualOffs))[tid];
    __syncthreads();
    
    // add up parts
    for (int order = 0; order <= max_order; order++)
    {
	shared.parts[tid] = tid < partCount ? autoc[(blockIdx.y * partCount + tid) * (max_order + 1) + order] : 0;
	__syncthreads();
	if (tid < 64 && blockDim.x > 64) shared.parts[tid] += shared.parts[tid + 64];
	__syncthreads();
	if (tid < 32) 
	{
	    if (blockDim.x > 32) shared.parts[tid] += shared.parts[tid + 32];
	    shared.parts[tid] += shared.parts[tid + 16];
	    shared.parts[tid] += shared.parts[tid + 8];
	    shared.parts[tid] += shared.parts[tid + 4];
	    shared.parts[tid] += shared.parts[tid + 2];
	    shared.parts[tid] += shared.parts[tid + 1];
	    if (tid == 0)
		shared.autoc[order] = shared.parts[0];
	}
    }
   
    if (tid < 32)
    {
	shared.gen0[tid] = shared.autoc[tid+1];
	shared.gen1[tid] = shared.autoc[tid+1];
	shared.ldr[tid] = 0.0f;

	float error = shared.autoc[0];
	for (int order = 0; order < max_order; order++)
	{
	    // Schur recursion
	    float reff = -shared.gen1[0] / error;
	    //if (tid == 0) shared.reff[order] = reff;
	    error += __fmul_rz(shared.gen1[0], reff);
	    if (tid < max_order - 1 - order)
	    {
		float g1 = shared.gen1[tid + 1] + __fmul_rz(reff, shared.gen0[tid]);
		float g0 = __fmul_rz(shared.gen1[tid + 1], reff) + shared.gen0[tid];
		shared.gen1[tid] = g1;
		shared.gen0[tid] = g0;
	    }

	    // Levinson-Durbin recursion
	    shared.ldr[tid] += (tid < order) * __fmul_rz(reff, shared.ldr[order - 1 - tid]) + (tid  == order) * reff;

	    // Quantization
	    //int precision = 13 - (shared.task.blocksize <= 2304) - (shared.task.blocksize <= 1152) - (shared.task.blocksize <= 576);
	    int precision = max(3, min(13 - (shared.task.blocksize <= 2304) - (shared.task.blocksize <= 1152) - (shared.task.blocksize <= 576), shared.task2.abits));
	    int taskNo = shared.task.residualOffs + order;
	    shared.bits[tid] = __mul24((33 - __clz(__float2int_rn(fabs(shared.ldr[tid]) * (1 << 15))) - precision), tid <= order);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int sh = max(0,min(15, 15 - shared.bits[0]));
	    
	    // reverse coefs
	    int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(-shared.ldr[order - tid] * (1 << sh))));
	    if (tid <= order)
		output[taskNo].coefs[tid] = coef;
	    if (tid == 0)
		output[taskNo].shift = sh;
	    shared.bits[tid] = __mul24(33 - __clz(coef ^ (coef >> 31)), tid <= order);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int cbits = shared.bits[0];
	    if (tid == 0)
		output[taskNo].cbits = cbits;
	}
    }
}

extern "C" __global__ void cudaComputeLPCLattice(
    encodeResidualTaskStruct *tasks,
    const int taskCount, // tasks per block
    const int *samples,
    const int precisions,
    const int max_order // should be <= 12
)
{
    __shared__ struct {
	volatile encodeResidualTaskStruct task;
	volatile float F[512];
	volatile float lpc[12][32];
	union {
	    volatile float tmp[256];
	    volatile int tmpi[256];
	};
    } shared;

    // fetch task data
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];
    __syncthreads();

    // F = samples; B = samples
    //int frameSize = shared.task.blocksize;
    int s1 = threadIdx.x < shared.task.blocksize ? samples[shared.task.samplesOffs + threadIdx.x] : 0;
    int s2 = threadIdx.x + 256 < shared.task.blocksize ? samples[shared.task.samplesOffs + threadIdx.x + 256] : 0;
    shared.tmpi[threadIdx.x] = s1|s2;
    __syncthreads();
    SUM256(shared.tmpi,threadIdx.x,|=);
    if (threadIdx.x == 0)
	shared.task.wbits = max(0,__ffs(shared.tmpi[0]) - 1);
    __syncthreads();
    if (threadIdx.x < taskCount)
	tasks[blockIdx.y * taskCount + threadIdx.x].wbits = shared.task.wbits;
    shared.tmpi[threadIdx.x] = (s1 ^ (s1 >> 31)) | (s2 ^ (s2 >> 31));
    __syncthreads();
    SUM256(shared.tmpi,threadIdx.x,|=);
    if (threadIdx.x == 0)
	shared.task.abits = 32 - __clz(shared.tmpi[0]) - shared.task.wbits;
    __syncthreads();
    s1 >>= shared.task.wbits;
    s2 >>= shared.task.wbits;
    shared.F[threadIdx.x] = s1;
    shared.F[threadIdx.x + 256] = s2;
    __syncthreads();

    for (int order = 1; order <= max_order; order++)
    {
	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
	float f1 = (threadIdx.x + order < shared.task.blocksize) * shared.F[threadIdx.x + order];
	float f2 = (threadIdx.x + 256 + order < shared.task.blocksize) * shared.F[threadIdx.x + 256 + order];
	s1 *= (threadIdx.x + order < shared.task.blocksize);
	s2 *= (threadIdx.x + 256 + order < shared.task.blocksize);

	// DEN = F(order+1:frameSize) * F(order+1:frameSize)' + B(1:frameSize-order) * B(1:frameSize-order)' (BURG)
	shared.tmp[threadIdx.x] = FSQR(f1) + FSQR(f2) + FSQR(s1) + FSQR(s2);
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	float DEN = shared.tmp[0] / 2;
	    //shared.PE[order-1] = shared.tmp[0] / 2 / (frameSize - order + 1);
	__syncthreads();

	shared.tmp[threadIdx.x] = f1 * s1 + f2 * s2;
	__syncthreads(); 
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	float reff = shared.tmp[0] / DEN;
	__syncthreads();

	// arp(order) = rc(order) = reff
	if (threadIdx.x == 0)
	    shared.lpc[order - 1][order - 1] = reff;
	    //shared.rc[order - 1] = shared.lpc[order - 1][order - 1] = reff;

	// Levinson-Durbin recursion
	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
	if (threadIdx.x < order - 1)
	    shared.lpc[order - 1][threadIdx.x] = shared.lpc[order - 2][threadIdx.x] - reff * shared.lpc[order - 2][order - 2 - threadIdx.x];

	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
	// F(order+1:frameSize) = F1
	if (threadIdx.x < shared.task.blocksize - order)
	    shared.F[order + threadIdx.x] -= reff * s1;
	if (threadIdx.x + 256 < shared.task.blocksize - order)
	    shared.F[order + threadIdx.x + 256] -= reff * s2;
	s1 -= reff * f1;
	s2 -= reff * f2;
	__syncthreads();
    }
    // Quantization
    for (int order = (threadIdx.x >> 5); order < max_order; order += 8)
    for (int precision = 0; precision < precisions; precision++)
    {
	int cn = threadIdx.x & 31;
	// get 15 bits of each coeff
	int coef = cn <= order ? __float2int_rn(shared.lpc[order][cn] * (1 << 15)) : 0;
	// remove sign bits
	shared.tmpi[threadIdx.x] = coef ^ (coef >> 31);
	// OR reduction
	SUM32(shared.tmpi,threadIdx.x,|=);
	// choose precision	
	//int cbits = max(3, min(10, 5 + (shared.task.abits >> 1))); //  - __float2int_rn(shared.PE[order - 1])
	int cbits = max(3, min(10, shared.task.abits)) - precision;// + precision); //  - __float2int_rn(shared.PE[order - 1])
	// calculate shift based on precision and number of leading zeroes in coeffs
	int shift = max(0,min(15, __clz(shared.tmpi[threadIdx.x & ~31]) - 18 + cbits));
	//if (shared.task.abits + 32 - __clz(order) < shift
	//int shift = max(0,min(15, (shared.task.abits >> 2) - 14 + __clz(shared.tmpi[threadIdx.x & ~31]) + ((32 - __clz(order))>>1)));
	// quantize coeffs with given shift
	coef = cn <= order ? max(-(1 << (cbits - 1)), min((1 << (cbits - 1)) -1, __float2int_rn(shared.lpc[order][order - cn] * (1 << shift)))) : 0;
	// error correction
	//shared.tmp[threadIdx.x] = (threadIdx.x != 0) * (shared.arp[threadIdx.x - 1]*(1 << shared.task.shift) - shared.task.coefs[threadIdx.x - 1]);
	//shared.task.coefs[threadIdx.x] = max(-(1 << (shared.task.cbits - 1)), min((1 << (shared.task.cbits - 1))-1, __float2int_rn((shared.arp[threadIdx.x]) * (1 << shared.task.shift) + shared.tmp[threadIdx.x])));
	// remove sign bits
	shared.tmpi[threadIdx.x] = coef ^ (coef >> 31);
	// OR reduction
	SUM32(shared.tmpi,threadIdx.x,|=);
	// calculate actual number of bits (+1 for sign)
	cbits = 1 + 32 - __clz(shared.tmpi[threadIdx.x & ~31]);

	// output shift, cbits and output coeffs
	int taskNo = taskCount * blockIdx.y + order + precision * max_order;
	if (cn == 0)
	    tasks[taskNo].shift = shift;
	if (cn == 0)
	    tasks[taskNo].cbits = cbits;
	if (cn <= order)
	    tasks[taskNo].coefs[cn] = coef;
    }
}

//extern "C" __global__ void cudaComputeLPCLattice512(
//    encodeResidualTaskStruct *tasks,
//    const int taskCount, // tasks per block
//    const int *samples,
//    const int frameSize, // <= 512
//    const int max_order // should be <= 32
//)
//{
//    __shared__ struct {
//	encodeResidualTaskStruct task;
//	float F[512];
//	float B[512];
//	float lpc[32][32];
//	volatile float tmp[512];
//	volatile float arp[32];
//	volatile float rc[32];
//	volatile int   bits[512];
//	volatile float f, b;
//    } shared;
//
//    // fetch task data
//    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
//	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];    
//    __syncthreads();
//
//    // F = samples; B = samples
//    shared.F[threadIdx.x] = threadIdx.x < frameSize ? samples[shared.task.samplesOffs + threadIdx.x] >> shared.task.wbits : 0.0f;
//    shared.B[threadIdx.x] = shared.F[threadIdx.x];
//    __syncthreads();
//
//    // DEN = F*F'
//    shared.tmp[threadIdx.x] = FSQR(shared.F[threadIdx.x]);
//    __syncthreads();
//    SUM512(shared.tmp,threadIdx.x,+=);
//    __syncthreads();
//    if (threadIdx.x == 0)
//	shared.f = shared.b = shared.tmp[0];
// //   if (threadIdx.x == 0)
//	//shared.PE[0] = DEN / frameSize;
//    __syncthreads();
//
//    for (int order = 1; order <= max_order; order++)
//    {
//	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
//	shared.tmp[threadIdx.x] = (threadIdx.x + order < frameSize) * shared.F[threadIdx.x + order] * shared.B[threadIdx.x];
//	__syncthreads(); 
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	
//	//float reff = shared.tmp[0] * rsqrtf(shared.b * shared.f); // Geometric lattice
//	float reff = shared.tmp[0] * 2 / (shared.b + shared.f); // Burg method
//	__syncthreads();
//
//	// Levinson-Durbin recursion
//	// arp(order) = rc(order) = reff
//	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
//	if (threadIdx.x == 32)
//	    shared.arp[order - 1] = shared.rc[order - 1] = reff;
//	if (threadIdx.x < 32)
//	    shared.arp[threadIdx.x] -= (threadIdx.x < order - 1) * __fmul_rz(reff, shared.arp[order - 2 - threadIdx.x]);
//
//	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
//	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
//	// F(order+1:frameSize) = F1
//	if (threadIdx.x < frameSize - order)
//	{
//	    float f;// = shared.F[threadIdx.x + order];
//	    shared.F[threadIdx.x + order] = (f = shared.F[threadIdx.x + order]) - reff * shared.B[threadIdx.x];
//	    shared.B[threadIdx.x] -= reff * f;
//	}
//	__syncthreads();
//
//	// f = F(order+1:frameSize) * F(order+1:frameSize)'
//	// b = B(1:frameSize-order) * B(1:frameSize-order)'
//	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.F[threadIdx.x + order]);
//	__syncthreads();
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	if (threadIdx.x == 0)
//	    shared.f = shared.tmp[0];
//	__syncthreads();
//
//	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.B[threadIdx.x]);
//	__syncthreads();
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	if (threadIdx.x == 0)
//	    shared.b = shared.tmp[0];
//	__syncthreads();
//
//	if (threadIdx.x < 32)
//	    shared.lpc[order - 1][threadIdx.x] = shared.arp[threadIdx.x];
//
//	//if (threadIdx.x == 0)
//	//    shared.PE[order] = (shared.b + shared.f) / 2 / (frameSize - order);
//	__syncthreads();
//    }
//    for (int order = 1 + (threadIdx.x >> 5); order <= max_order; order += 16)
//    {
//	// Quantization
//	int cn = threadIdx.x & 31;
//	int precision = 10 - (order > 8) - min(2, shared.task.wbits);
//	int taskNo = taskCount * blockIdx.y + order - 1;
//	shared.bits[threadIdx.x] = __mul24((33 - __clz(__float2int_rn(fabs(shared.lpc[order - 1][cn]) * (1 << 15))) - precision), cn < order);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
//	int sh = max(0,min(15, 15 - shared.bits[threadIdx.x - cn]));
//            
//	// reverse coefs
//	int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(shared.lpc[order - 1][order - 1 - cn] * (1 << sh))));
//	if (cn < order)
//	    tasks[taskNo].coefs[cn] = coef;
//	if (cn == 0)
//	    tasks[taskNo].shift = sh;
//	shared.bits[threadIdx.x] = __mul24(33 - max(__clz(coef),__clz(-1 ^ coef)), cn < order);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
//	int cbits = shared.bits[threadIdx.x - cn];
//	if (cn == 0)
//	    tasks[taskNo].cbits = cbits;
//    }
//}

// blockDim.x == 32
// blockDim.y == 8
extern "C" __global__ void cudaEstimateResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks,
    int max_order,
    int frameSize,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	int data[32*9];
	volatile int residual[32*8];
	encodeResidualTaskStruct task[8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (threadIdx.x < 16)
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[blockIdx.y * blockDim.y + threadIdx.y]))[threadIdx.x];
    __syncthreads();
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(frameSize - pos, partSize + max_order);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task[0].samplesOffs + pos + tid] >> shared.task[0].wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task[0].samplesOffs + pos + tid + partSize] >> shared.task[0].wbits : 0;
    const int residualLen = max(0,min(frameSize - pos - shared.task[threadIdx.y].residualOrder, partSize));

    __syncthreads();

    shared.residual[tid] = 0;
    shared.task[threadIdx.y].coefs[threadIdx.x] = threadIdx.x < max_order ? tasks[blockIdx.y * blockDim.y + threadIdx.y].coefs[threadIdx.x] : 0;

    for (int i = blockDim.y * (shared.task[threadIdx.y].type == Verbatim); i < blockDim.y; i++) // += 32
    {
	int ptr = threadIdx.x + (i<<5);
	// compute residual
	int sum = 0;
	int c = 0;
	for (c = 0; c < shared.task[threadIdx.y].residualOrder; c++)
	    sum += __mul24(shared.data[ptr + c], shared.task[threadIdx.y].coefs[c]);
	sum = shared.data[ptr + c] - (sum >> shared.task[threadIdx.y].shift);
	shared.residual[tid] += __mul24(ptr < residualLen, min(0x7fffff,(sum << 1) ^ (sum >> 31)));
    }

    // enable this line when using blockDim.x == 64
    //__syncthreads(); if (threadIdx.x < 32) shared.residual[tid] += shared.residual[tid + 32]; __syncthreads();
    shared.residual[tid] += shared.residual[tid + 16];
    shared.residual[tid] += shared.residual[tid + 8];
    shared.residual[tid] += shared.residual[tid + 4];
    shared.residual[tid] += shared.residual[tid + 2];
    shared.residual[tid] += shared.residual[tid + 1];

    // rice parameter search
    shared.residual[tid] = (shared.task[threadIdx.y].type != Constant || shared.residual[threadIdx.y * blockDim.x] != 0) *
	(__mul24(threadIdx.x >= 15, 0x7fffff) + residualLen * (threadIdx.x + 1) + ((shared.residual[threadIdx.y * blockDim.x] - (residualLen >> 1)) >> threadIdx.x));
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 8]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 4]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 2]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 1]);
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid];
}

#define BEST_INDEX(a,b) ((a) + ((b) - (a)) * (shared.length[b] < shared.length[a]))

extern "C" __global__ void cudaChooseBestMethod(
    encodeResidualTaskStruct *tasks,
    int *residual,
    int partCount, // <= blockDim.y (256)
    int taskCount
    )
{
    __shared__ struct {
	volatile int index[128];
	volatile int partLen[512];
	int length[256];
	volatile encodeResidualTaskStruct task[16];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    
    if (tid < 256) shared.length[tid] = 0x7fffffff;
    for (int task = 0; task < taskCount; task += blockDim.y)
	if (task + threadIdx.y < taskCount)
	{
	    // fetch task data
	    ((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(tasks + task + threadIdx.y + taskCount * blockIdx.y))[threadIdx.x];

	    int sum = 0;
	    for (int pos = 0; pos < partCount; pos += blockDim.x)
		sum += (pos + threadIdx.x < partCount ? residual[pos + threadIdx.x + 64 * (task + threadIdx.y + taskCount * blockIdx.y)] : 0);
	    shared.partLen[tid] = sum;

	    // length sum: reduction in shared mem
	    shared.partLen[tid] += shared.partLen[tid + 16];
	    shared.partLen[tid] += shared.partLen[tid + 8];
	    shared.partLen[tid] += shared.partLen[tid + 4];
	    shared.partLen[tid] += shared.partLen[tid + 2];
	    shared.partLen[tid] += shared.partLen[tid + 1];
	    // return sum
	    if (threadIdx.x == 0)
	    {
		int obits = shared.task[threadIdx.y].obits - shared.task[threadIdx.y].wbits;
		shared.length[task + threadIdx.y] =
		    min(obits * shared.task[threadIdx.y].blocksize,
			shared.task[threadIdx.y].type == Fixed ? shared.task[threadIdx.y].residualOrder * obits + 6 + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == LPC ? shared.task[threadIdx.y].residualOrder * obits + 4 + 5 + shared.task[threadIdx.y].residualOrder * shared.task[threadIdx.y].cbits + 6 + (4 * partCount/2)/* << porder */ + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == Constant ? obits * (1 + shared.task[threadIdx.y].blocksize * (shared.partLen[threadIdx.y * 32] != 0)) : 
			obits * shared.task[threadIdx.y].blocksize);
	    }
	}
    //shared.index[threadIdx.x] = threadIdx.x;
    //shared.length[threadIdx.x] = (threadIdx.x < taskCount) ? tasks[threadIdx.x + taskCount * blockIdx.y].size : 0x7fffffff;

    __syncthreads();

    //if (tid < 128) shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 128]); __syncthreads();
    if (tid < 128) shared.index[tid] = BEST_INDEX(tid, tid + 128); __syncthreads();
    if (tid < 64) shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 64]); __syncthreads();
    if (tid < 32) 
    {
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 32]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 16]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 8]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 4]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 2]);
	shared.index[tid] = BEST_INDEX(shared.index[tid], shared.index[tid + 1]);
    }
    __syncthreads();
 //   if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	//((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y + shared.index[0]))[threadIdx.x];
    if (tid == 0)
	tasks[taskCount * blockIdx.y].best_index = taskCount * blockIdx.y + shared.index[0];
    if (tid < taskCount)
	tasks[tid + taskCount * blockIdx.y].size = shared.length[tid];
}

extern "C" __global__ void cudaCopyBestMethod(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index;
    } shared;
    if (threadIdx.x == 0)
	shared.best_index = tasks[count * blockIdx.y].best_index;
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.best_index))[threadIdx.x];
}

extern "C" __global__ void cudaCopyBestMethodStereo(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index[4];
	int best_size[4];
	int lr_index[2];
    } shared;
    if (threadIdx.x < 4)
	shared.best_index[threadIdx.x] = tasks[count * (blockIdx.y * 4 + threadIdx.x)].best_index;
    if (threadIdx.x < 4)
	shared.best_size[threadIdx.x] = tasks[shared.best_index[threadIdx.x]].size;
    __syncthreads();
    if (threadIdx.x == 0)
    {
	int bitsBest = 0x7fffffff;
	if (bitsBest > shared.best_size[2] + shared.best_size[3]) // MidSide
	{
	    bitsBest = shared.best_size[2] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[2];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[3] + shared.best_size[1]) // RightSide
	{
	    bitsBest = shared.best_size[3] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[3];
	    shared.lr_index[1] = shared.best_index[1];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[3]) // LeftSide
	{
	    bitsBest = shared.best_size[0] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[1]) // LeftRight
	{
	    bitsBest = shared.best_size[0] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[1];
	}
    }
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.lr_index[0]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y].residualOffs = tasks[shared.best_index[0]].residualOffs;
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y + 1))[threadIdx.x] = ((int*)(tasks + shared.lr_index[1]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y + 1].residualOffs = tasks[shared.best_index[1]].residualOffs;
}

extern "C" __global__ void cudaEncodeResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks
    )
{
    __shared__ struct {
	int data[256 + 32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x;
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    __syncthreads();
    const int partSize = blockDim.x;
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(shared.task.blocksize - pos, partSize + shared.task.residualOrder);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] >> shared.task.wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task.samplesOffs + pos + tid + partSize] >> shared.task.wbits : 0;
    const int residualLen = max(0,min(shared.task.blocksize - pos - shared.task.residualOrder, partSize));

    __syncthreads();
    
    // compute residual
    int sum = 0;
    for (int c = 0; c < shared.task.residualOrder; c++)
	sum += __mul24(shared.data[tid + c], shared.task.coefs[c]);
    if (tid < residualLen)
	output[shared.task.residualOffs + pos + tid] = shared.data[tid + shared.task.residualOrder] - (sum >> shared.task.shift);
}

extern "C" __global__ void cudaCalcPartition(
    int* partition_lengths,
    int* residual,
    encodeResidualTaskStruct *tasks,
    int max_porder // <= 8
    )
{
    __shared__ struct {
	int data[256];
	int length[256];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int psize = (shared.task.blocksize >> max_porder); // 18
    const int parts_per_block = 256 / psize; // 14
    const int parts = min(parts_per_block, (1 << max_porder) - blockIdx.x * parts_per_block);

    // fetch residual
    shared.data[tid] = ((blockIdx.x != 0 || tid >= shared.task.residualOrder) && tid < parts * psize) ? residual[shared.task.residualOffs + blockIdx.x * psize * parts_per_block + tid - shared.task.residualOrder] : 0;
    // convert to unsigned
    shared.data[tid] = (shared.data[tid] << 1) ^ (shared.data[tid] >> 31);
    __syncthreads();

    // calc number of unary bits for each residual part with each rice paramater
    shared.length[tid] = (psize - shared.task.residualOrder * (threadIdx.y + blockIdx.x == 0)) * (threadIdx.x + 1);
    if (threadIdx.y < parts)
	for (int i = 0; i < psize; i++)
	// for part (threadIdx.y) with this rice paramater (threadIdx.x)
	    shared.length[tid] = min(0xfffff, shared.length[tid] + (shared.data[threadIdx.y * psize + i] >> threadIdx.x));
    __syncthreads();

    // output length (transposed: k is now threadIdx.y)
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x < parts)
	partition_lengths[pos + blockIdx.x * parts_per_block + threadIdx.x] = shared.length[threadIdx.y + (threadIdx.x << 4)];
}

extern "C" __global__ void cudaCalcLargePartition(
    int* partition_lengths,
    int* residual,
    encodeResidualTaskStruct *tasks,
    int max_porder // <= 8
    )
{
    __shared__ struct {
	int data[256];
	int length[256];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int psize = (shared.task.blocksize >> max_porder); // 18

    shared.length[tid] = 0;
    for (int pos = 0; pos < psize; pos += 256)
    {
	// fetch residual
	shared.data[tid] = ((blockIdx.x != 0 || pos + tid >= shared.task.residualOrder) && pos + tid < psize) ? residual[shared.task.residualOffs + blockIdx.x * psize + pos + tid - shared.task.residualOrder] : 0;
	// convert to unsigned
	shared.data[tid] = (shared.data[tid] << 1) ^ (shared.data[tid] >> 31);
	__syncthreads();

	// calc number of unary bits for each residual sample with each rice paramater
	for (int i = 0; i < 256; i += 16)
	    // for sample (i + threadIdx.x) with this rice paramater (threadIdx.y)
	    shared.length[tid] = min(0xfffff, shared.length[tid] + (shared.data[i + threadIdx.x] >> threadIdx.y));
	__syncthreads();
    }
    __syncthreads();
    SUM16(shared.length,tid,+=);
    __syncthreads();

    // output length
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x == 0)
	partition_lengths[pos + blockIdx.x] = shared.length[tid] + (psize - shared.task.residualOrder * (blockIdx.x == 0)) * (threadIdx.y + 1);
}

// Sums partition lengths for a certain k == blockIdx.x
// Requires 256 threads
extern "C" __global__ void cudaSumPartition(
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	int data[512];
    } shared;

    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (blockIdx.x << (max_porder + 1));

    // fetch residual
    shared.data[threadIdx.x] = threadIdx.x < (1 << max_porder) ? partition_lengths[pos + threadIdx.x] : 0;
    __syncthreads();
    for (int porder = max_porder - 1; porder >= 0; porder--)
    {
	const int in_pos = (2 << max_porder) - (4 << porder);
	const int out_pos = (2 << max_porder) - (2 << porder);
	if (threadIdx.x < (1 << porder)) shared.data[out_pos + threadIdx.x] = shared.data[in_pos + (threadIdx.x << 1)] + shared.data[in_pos + (threadIdx.x << 1) + 1];
	__syncthreads();
    }
    if (threadIdx.x < (1 << max_porder))
	partition_lengths[pos + (1 << max_porder) + threadIdx.x] = shared.data[(1 << max_porder) + threadIdx.x];
}

// Finds optimal rice parameter for up to 16 partitions at a time.
// Requires 16x16 threads
extern "C" __global__ void cudaFindRiceParameter(
    int* output,
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	int length[256];
	int tmp[256];
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    const int parts = min(16, 2 << max_porder);
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));

    // read length for 16 partitions
    shared.length[tid] = (threadIdx.y <= 14 && threadIdx.x < parts) ? partition_lengths[pos + blockIdx.x * 16 + threadIdx.x] : 0xffffff;
    __syncthreads();
    // transpose
    shared.tmp[tid] = threadIdx.y + (threadIdx.x << 4);
    // find best rice parameter
    shared.tmp[tid] = BEST_INDEX(shared.tmp[tid], shared.tmp[tid + 8]);
    shared.tmp[tid] = BEST_INDEX(shared.tmp[tid], shared.tmp[tid + 4]);
    shared.tmp[tid] = BEST_INDEX(shared.tmp[tid], shared.tmp[tid + 2]);
    shared.tmp[tid] = BEST_INDEX(shared.tmp[tid], shared.tmp[tid + 1]);
    __syncthreads();
    // output rice parameter
    if (threadIdx.x == 0 && threadIdx.y < parts)
	output[(blockIdx.y << (max_porder + 2)) + blockIdx.x * parts + threadIdx.y] = shared.tmp[tid] >> 4;
    // output length
    if (threadIdx.x == 0 && threadIdx.y < parts)
	output[(blockIdx.y << (max_porder + 2)) + (1 << (max_porder + 1)) + blockIdx.x * parts + threadIdx.y] = shared.length[shared.tmp[tid]];
}

#endif
