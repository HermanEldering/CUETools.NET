/**
 * CUETools.FlaCuda: FLAC audio encoder using CUDA
 * Copyright (c) 2009 Gregory S. Chudov
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#ifndef _FLACUDA_KERNEL_H_
#define _FLACUDA_KERNEL_H_

typedef struct
{
    int samplesOffs;
    int windowOffs;
    int residualOffs;
    int blocksize;
    int reserved[12];
} computeAutocorTaskStruct;

typedef enum
{
    Constant = 0,
    Verbatim = 1,
    Fixed = 8,
    LPC = 32
} SubframeType;

typedef struct
{
    int residualOrder; // <= 32
    int samplesOffs;
    int shift;
    int cbits;
    int size;
    int type;
    int obits;
    int blocksize;
    int best_index;
    int channel;
    int residualOffs;
    int wbits;
    int abits;
    int porder;
    int reserved[2];
    int coefs[32]; // fixme: should be short?
} encodeResidualTaskStruct;

#define SUM16(buf,tid,op)   buf[tid] op buf[tid + 8]; buf[tid] op buf[tid + 4]; buf[tid] op buf[tid + 2]; buf[tid] op buf[tid + 1];
#define SUM32(buf,tid,op)   buf[tid] op buf[tid + 16]; SUM16(buf,tid,op)
#define SUM64(buf,tid,op)   if (tid < 32) buf[tid] op buf[tid + 32]; __syncthreads(); if (tid < 32) { SUM32(buf,tid,op) }
#define SUM128(buf,tid,op)  if (tid < 64) buf[tid] op buf[tid + 64]; __syncthreads(); SUM64(buf,tid,op)
#define SUM256(buf,tid,op)  if (tid < 128) buf[tid] op buf[tid + 128]; __syncthreads(); SUM128(buf,tid,op)
#define SUM512(buf,tid,op)  if (tid < 256) buf[tid] op buf[tid + 256]; __syncthreads(); SUM256(buf,tid,op)

#define FSQR(s) ((s)*(s))

extern "C" __global__ void cudaStereoDecorr(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
	samples[2 * offset + pos] = (s.x + s.y) >> 1;
	samples[3 * offset + pos] = s.x - s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr2(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr(
    int *samples,
    short *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
	samples[blockIdx.y * offset + pos] = src[pos * gridDim.y + blockIdx.y];
}

extern "C" __global__ void cudaFindWastedBits(
    encodeResidualTaskStruct *tasks,
    int *samples,
    int tasksPerChannel,
    int blocksize
)
{
    __shared__ struct {
	volatile int wbits[256];
	volatile int abits[256];
	encodeResidualTaskStruct task;
    } shared;

    if (threadIdx.x < 16)
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.x * tasksPerChannel]))[threadIdx.x];
    shared.wbits[threadIdx.x] = 0;
    shared.abits[threadIdx.x] = 0;
    __syncthreads();

    for (int pos = 0; pos < blocksize; pos += blockDim.x)
    {
	int smp = pos + threadIdx.x < blocksize ? samples[shared.task.samplesOffs + pos + threadIdx.x] : 0;
	shared.wbits[threadIdx.x] |= smp;
	shared.abits[threadIdx.x] |= smp ^ (smp >> 31);
    }
    __syncthreads();
    SUM256(shared.wbits, threadIdx.x, |=);
    SUM256(shared.abits, threadIdx.x, |=);
    if (threadIdx.x == 0)
	shared.task.wbits = max(0,__ffs(shared.wbits[0]) - 1);
    if (threadIdx.x == 0)
	shared.task.abits = 32 - __clz(shared.abits[0]) - shared.task.wbits;
    __syncthreads();

    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].wbits = shared.task.wbits;
    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].abits = shared.task.abits;
}

extern "C" __global__ void cudaComputeAutocor(
    float *output,
    const int *samples,
    const float *window,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int frameSize,
    int partSize // should be <= 2*blockDim - max_order
)
{
    __shared__ struct {
	float data[512];
	volatile float product[256];
	computeAutocorTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y * 32);
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();

    // fetch samples
    {
	const int pos = blockIdx.x * partSize;
	const int dataLen = min(frameSize - pos, partSize + max_order);

	shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] * window[shared.task.windowOffs + pos + tid]: 0.0f;
	shared.data[tid + 256] = tid + 256 < dataLen ? samples[shared.task.samplesOffs + pos + tid + 256] * window[shared.task.windowOffs + pos + tid + 256]: 0.0f;
    }
    __syncthreads();

    for (int lag = threadIdx.y; lag <= max_order; lag += 8)
    {
        const int productLen = min(frameSize - blockIdx.x * partSize - lag, partSize);
	shared.product[tid] = 0.0;
	for (int ptr = threadIdx.x; ptr < productLen + threadIdx.x; ptr += 128)
	    shared.product[tid] += ((ptr < productLen) * shared.data[ptr] * shared.data[ptr + lag]
				 + (ptr + 32 < productLen) * shared.data[ptr + 32] * shared.data[ptr + 32 + lag])
				 + ((ptr + 64 < productLen) * shared.data[ptr + 64] * shared.data[ptr + 64 + lag]
				 + (ptr + 96 < productLen) * shared.data[ptr + 96] * shared.data[ptr + 96 + lag]);
	// product sum: reduction in shared mem
	//shared.product[tid] += shared.product[tid + 16];
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 16]) + (shared.product[tid + 8] + shared.product[tid + 24]);
	shared.product[tid] = (shared.product[tid] + shared.product[tid + 4]) + (shared.product[tid + 2] + shared.product[tid + 6]);
	// return results
	if (threadIdx.x == 0)
	    output[(blockIdx.x + blockIdx.y * gridDim.x) * (max_order + 1) + lag] = shared.product[tid] + shared.product[tid + 1];
    }
}

extern "C" __global__ void cudaComputeLPC(
    encodeResidualTaskStruct *output,
    float*autoc,
    computeAutocorTaskStruct *tasks,
    int max_order, // should be <= 32
    int partCount // should be <= blockDim?
)
{
    __shared__ struct {
	computeAutocorTaskStruct task;
	encodeResidualTaskStruct task2;
	volatile float ldr[32];
	volatile int   bits[32];
	volatile float autoc[33];
	volatile float gen0[32];
	volatile float gen1[32];
	volatile float parts[128];
	//volatile float reff[32];
	//int   cbits;
    } shared;
    const int tid = threadIdx.x;
    
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y))[tid];
    __syncthreads();
    if (tid < sizeof(shared.task2) / sizeof(int))
	((int*)&shared.task2)[tid] = ((int*)(output + shared.task.residualOffs))[tid];
    __syncthreads();
    
    // add up parts
    for (int order = 0; order <= max_order; order++)
    {
	shared.parts[tid] = tid < partCount ? autoc[(blockIdx.y * partCount + tid) * (max_order + 1) + order] : 0;
	__syncthreads();
	if (tid < 64 && blockDim.x > 64) shared.parts[tid] += shared.parts[tid + 64];
	__syncthreads();
	if (tid < 32) 
	{
	    if (blockDim.x > 32) shared.parts[tid] += shared.parts[tid + 32];
	    shared.parts[tid] += shared.parts[tid + 16];
	    shared.parts[tid] += shared.parts[tid + 8];
	    shared.parts[tid] += shared.parts[tid + 4];
	    shared.parts[tid] += shared.parts[tid + 2];
	    shared.parts[tid] += shared.parts[tid + 1];
	    if (tid == 0)
		shared.autoc[order] = shared.parts[0];
	}
    }
   
    if (tid < 32)
    {
	shared.gen0[tid] = shared.autoc[tid+1];
	shared.gen1[tid] = shared.autoc[tid+1];
	shared.ldr[tid] = 0.0f;

	float error = shared.autoc[0];
	for (int order = 0; order < max_order; order++)
	{
	    // Schur recursion
	    float reff = -shared.gen1[0] / error;
	    //if (tid == 0) shared.reff[order] = reff;
	    error += __fmul_rz(shared.gen1[0], reff);
	    if (tid < max_order - 1 - order)
	    {
		float g1 = shared.gen1[tid + 1] + __fmul_rz(reff, shared.gen0[tid]);
		float g0 = __fmul_rz(shared.gen1[tid + 1], reff) + shared.gen0[tid];
		shared.gen1[tid] = g1;
		shared.gen0[tid] = g0;
	    }

	    // Levinson-Durbin recursion
	    shared.ldr[tid] += (tid < order) * __fmul_rz(reff, shared.ldr[order - 1 - tid]) + (tid  == order) * reff;

	    // Quantization
	    //int precision = 13 - (shared.task.blocksize <= 2304) - (shared.task.blocksize <= 1152) - (shared.task.blocksize <= 576);
	    int precision = max(3, min(13 - (shared.task.blocksize <= 2304) - (shared.task.blocksize <= 1152) - (shared.task.blocksize <= 576), shared.task2.abits));
	    int taskNo = shared.task.residualOffs + order;
	    shared.bits[tid] = __mul24((33 - __clz(__float2int_rn(fabs(shared.ldr[tid]) * (1 << 15))) - precision), tid <= order);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int sh = max(0,min(15, 15 - shared.bits[0]));
	    
	    // reverse coefs
	    int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(-shared.ldr[order - tid] * (1 << sh))));
	    if (tid <= order)
		output[taskNo].coefs[tid] = coef;
	    if (tid == 0)
		output[taskNo].shift = sh;
	    shared.bits[tid] = __mul24(33 - __clz(coef ^ (coef >> 31)), tid <= order);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 16]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 8]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 4]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 2]);
	    shared.bits[tid] = max(shared.bits[tid], shared.bits[tid + 1]);
	    int cbits = shared.bits[0];
	    if (tid == 0)
		output[taskNo].cbits = cbits;
	}
    }
}

extern "C" __global__ void cudaComputeLPCLattice(
    encodeResidualTaskStruct *tasks,
    const int taskCount, // tasks per block
    const int *samples,
    const int precisions,
    const int max_order // should be <= 12
)
{
    __shared__ struct {
	volatile encodeResidualTaskStruct task;
	volatile float F[512];
	volatile float lpc[12][32];
	union {
	    volatile float tmp[256];
	    volatile int tmpi[256];
	};
    } shared;

    // fetch task data
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];
    __syncthreads();

    // F = samples; B = samples
    //int frameSize = shared.task.blocksize;
    int s1 = threadIdx.x < shared.task.blocksize ? samples[shared.task.samplesOffs + threadIdx.x] : 0;
    int s2 = threadIdx.x + 256 < shared.task.blocksize ? samples[shared.task.samplesOffs + threadIdx.x + 256] : 0;
    shared.tmpi[threadIdx.x] = s1|s2;
    __syncthreads();
    SUM256(shared.tmpi,threadIdx.x,|=);
    if (threadIdx.x == 0)
	shared.task.wbits = max(0,__ffs(shared.tmpi[0]) - 1);
    __syncthreads();
    if (threadIdx.x < taskCount)
	tasks[blockIdx.y * taskCount + threadIdx.x].wbits = shared.task.wbits;
    shared.tmpi[threadIdx.x] = (s1 ^ (s1 >> 31)) | (s2 ^ (s2 >> 31));
    __syncthreads();
    SUM256(shared.tmpi,threadIdx.x,|=);
    if (threadIdx.x == 0)
	shared.task.abits = 32 - __clz(shared.tmpi[0]) - shared.task.wbits;
    __syncthreads();
    s1 >>= shared.task.wbits;
    s2 >>= shared.task.wbits;
    shared.F[threadIdx.x] = s1;
    shared.F[threadIdx.x + 256] = s2;
    __syncthreads();

    for (int order = 1; order <= max_order; order++)
    {
	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
	float f1 = (threadIdx.x + order < shared.task.blocksize) * shared.F[threadIdx.x + order];
	float f2 = (threadIdx.x + 256 + order < shared.task.blocksize) * shared.F[threadIdx.x + 256 + order];
	s1 *= (threadIdx.x + order < shared.task.blocksize);
	s2 *= (threadIdx.x + 256 + order < shared.task.blocksize);

	// DEN = F(order+1:frameSize) * F(order+1:frameSize)' + B(1:frameSize-order) * B(1:frameSize-order)' (BURG)
	shared.tmp[threadIdx.x] = FSQR(f1) + FSQR(f2) + FSQR(s1) + FSQR(s2);
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	float DEN = shared.tmp[0] / 2;
	    //shared.PE[order-1] = shared.tmp[0] / 2 / (frameSize - order + 1);
	__syncthreads();

	shared.tmp[threadIdx.x] = f1 * s1 + f2 * s2;
	__syncthreads(); 
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	float reff = shared.tmp[0] / DEN;
	__syncthreads();

	// arp(order) = rc(order) = reff
	if (threadIdx.x == 0)
	    shared.lpc[order - 1][order - 1] = reff;
	    //shared.rc[order - 1] = shared.lpc[order - 1][order - 1] = reff;

	// Levinson-Durbin recursion
	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
	if (threadIdx.x < order - 1)
	    shared.lpc[order - 1][threadIdx.x] = shared.lpc[order - 2][threadIdx.x] - reff * shared.lpc[order - 2][order - 2 - threadIdx.x];

	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
	// F(order+1:frameSize) = F1
	if (threadIdx.x < shared.task.blocksize - order)
	    shared.F[order + threadIdx.x] -= reff * s1;
	if (threadIdx.x + 256 < shared.task.blocksize - order)
	    shared.F[order + threadIdx.x + 256] -= reff * s2;
	s1 -= reff * f1;
	s2 -= reff * f2;
	__syncthreads();
    }
    // Quantization
    for (int order = (threadIdx.x >> 5); order < max_order; order += 8)
    for (int precision = 0; precision < precisions; precision++)
    {
	int cn = threadIdx.x & 31;
	// get 15 bits of each coeff
	int coef = cn <= order ? __float2int_rn(shared.lpc[order][cn] * (1 << 15)) : 0;
	// remove sign bits
	shared.tmpi[threadIdx.x] = coef ^ (coef >> 31);
	// OR reduction
	SUM32(shared.tmpi,threadIdx.x,|=);
	// choose precision	
	//int cbits = max(3, min(10, 5 + (shared.task.abits >> 1))); //  - __float2int_rn(shared.PE[order - 1])
	int cbits = max(3, min(10, shared.task.abits)) - precision;// + precision); //  - __float2int_rn(shared.PE[order - 1])
	// calculate shift based on precision and number of leading zeroes in coeffs
	int shift = max(0,min(15, __clz(shared.tmpi[threadIdx.x & ~31]) - 18 + cbits));
	//if (shared.task.abits + 32 - __clz(order) < shift
	//int shift = max(0,min(15, (shared.task.abits >> 2) - 14 + __clz(shared.tmpi[threadIdx.x & ~31]) + ((32 - __clz(order))>>1)));
	// quantize coeffs with given shift
	coef = cn <= order ? max(-(1 << (cbits - 1)), min((1 << (cbits - 1)) -1, __float2int_rn(shared.lpc[order][order - cn] * (1 << shift)))) : 0;
	// error correction
	//shared.tmp[threadIdx.x] = (threadIdx.x != 0) * (shared.arp[threadIdx.x - 1]*(1 << shared.task.shift) - shared.task.coefs[threadIdx.x - 1]);
	//shared.task.coefs[threadIdx.x] = max(-(1 << (shared.task.cbits - 1)), min((1 << (shared.task.cbits - 1))-1, __float2int_rn((shared.arp[threadIdx.x]) * (1 << shared.task.shift) + shared.tmp[threadIdx.x])));
	// remove sign bits
	shared.tmpi[threadIdx.x] = coef ^ (coef >> 31);
	// OR reduction
	SUM32(shared.tmpi,threadIdx.x,|=);
	// calculate actual number of bits (+1 for sign)
	cbits = 1 + 32 - __clz(shared.tmpi[threadIdx.x & ~31]);

	// output shift, cbits and output coeffs
	int taskNo = taskCount * blockIdx.y + order + precision * max_order;
	if (cn == 0)
	    tasks[taskNo].shift = shift;
	if (cn == 0)
	    tasks[taskNo].cbits = cbits;
	if (cn <= order)
	    tasks[taskNo].coefs[cn] = coef;
    }
}

//extern "C" __global__ void cudaComputeLPCLattice512(
//    encodeResidualTaskStruct *tasks,
//    const int taskCount, // tasks per block
//    const int *samples,
//    const int frameSize, // <= 512
//    const int max_order // should be <= 32
//)
//{
//    __shared__ struct {
//	encodeResidualTaskStruct task;
//	float F[512];
//	float B[512];
//	float lpc[32][32];
//	volatile float tmp[512];
//	volatile float arp[32];
//	volatile float rc[32];
//	volatile int   bits[512];
//	volatile float f, b;
//    } shared;
//
//    // fetch task data
//    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
//	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];    
//    __syncthreads();
//
//    // F = samples; B = samples
//    shared.F[threadIdx.x] = threadIdx.x < frameSize ? samples[shared.task.samplesOffs + threadIdx.x] >> shared.task.wbits : 0.0f;
//    shared.B[threadIdx.x] = shared.F[threadIdx.x];
//    __syncthreads();
//
//    // DEN = F*F'
//    shared.tmp[threadIdx.x] = FSQR(shared.F[threadIdx.x]);
//    __syncthreads();
//    SUM512(shared.tmp,threadIdx.x,+=);
//    __syncthreads();
//    if (threadIdx.x == 0)
//	shared.f = shared.b = shared.tmp[0];
// //   if (threadIdx.x == 0)
//	//shared.PE[0] = DEN / frameSize;
//    __syncthreads();
//
//    for (int order = 1; order <= max_order; order++)
//    {
//	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
//	shared.tmp[threadIdx.x] = (threadIdx.x + order < frameSize) * shared.F[threadIdx.x + order] * shared.B[threadIdx.x];
//	__syncthreads(); 
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	
//	//float reff = shared.tmp[0] * rsqrtf(shared.b * shared.f); // Geometric lattice
//	float reff = shared.tmp[0] * 2 / (shared.b + shared.f); // Burg method
//	__syncthreads();
//
//	// Levinson-Durbin recursion
//	// arp(order) = rc(order) = reff
//	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
//	if (threadIdx.x == 32)
//	    shared.arp[order - 1] = shared.rc[order - 1] = reff;
//	if (threadIdx.x < 32)
//	    shared.arp[threadIdx.x] -= (threadIdx.x < order - 1) * __fmul_rz(reff, shared.arp[order - 2 - threadIdx.x]);
//
//	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
//	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
//	// F(order+1:frameSize) = F1
//	if (threadIdx.x < frameSize - order)
//	{
//	    float f;// = shared.F[threadIdx.x + order];
//	    shared.F[threadIdx.x + order] = (f = shared.F[threadIdx.x + order]) - reff * shared.B[threadIdx.x];
//	    shared.B[threadIdx.x] -= reff * f;
//	}
//	__syncthreads();
//
//	// f = F(order+1:frameSize) * F(order+1:frameSize)'
//	// b = B(1:frameSize-order) * B(1:frameSize-order)'
//	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.F[threadIdx.x + order]);
//	__syncthreads();
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	if (threadIdx.x == 0)
//	    shared.f = shared.tmp[0];
//	__syncthreads();
//
//	shared.tmp[threadIdx.x] = (threadIdx.x < frameSize - order) * FSQR(shared.B[threadIdx.x]);
//	__syncthreads();
//	SUM512(shared.tmp, threadIdx.x,+=);
//	__syncthreads();
//	if (threadIdx.x == 0)
//	    shared.b = shared.tmp[0];
//	__syncthreads();
//
//	if (threadIdx.x < 32)
//	    shared.lpc[order - 1][threadIdx.x] = shared.arp[threadIdx.x];
//
//	//if (threadIdx.x == 0)
//	//    shared.PE[order] = (shared.b + shared.f) / 2 / (frameSize - order);
//	__syncthreads();
//    }
//    for (int order = 1 + (threadIdx.x >> 5); order <= max_order; order += 16)
//    {
//	// Quantization
//	int cn = threadIdx.x & 31;
//	int precision = 10 - (order > 8) - min(2, shared.task.wbits);
//	int taskNo = taskCount * blockIdx.y + order - 1;
//	shared.bits[threadIdx.x] = __mul24((33 - __clz(__float2int_rn(fabs(shared.lpc[order - 1][cn]) * (1 << 15))) - precision), cn < order);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
//	int sh = max(0,min(15, 15 - shared.bits[threadIdx.x - cn]));
//            
//	// reverse coefs
//	int coef = max(-(1 << precision),min((1 << precision)-1,__float2int_rn(shared.lpc[order - 1][order - 1 - cn] * (1 << sh))));
//	if (cn < order)
//	    tasks[taskNo].coefs[cn] = coef;
//	if (cn == 0)
//	    tasks[taskNo].shift = sh;
//	shared.bits[threadIdx.x] = __mul24(33 - max(__clz(coef),__clz(-1 ^ coef)), cn < order);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 16]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 8]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 4]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 2]);
//	shared.bits[threadIdx.x] = max(shared.bits[threadIdx.x], shared.bits[threadIdx.x + 1]);
//	int cbits = shared.bits[threadIdx.x - cn];
//	if (cn == 0)
//	    tasks[taskNo].cbits = cbits;
//    }
//}

// blockDim.x == 32
// blockDim.y == 8
extern "C" __global__ void cudaEstimateResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks,
    int max_order,
    int frameSize,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	int data[32*9];
	volatile int residual[32*8];
	encodeResidualTaskStruct task[8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (threadIdx.x < 16)
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[blockIdx.y * blockDim.y + threadIdx.y]))[threadIdx.x];
    __syncthreads();
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(frameSize - pos, partSize + max_order);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task[0].samplesOffs + pos + tid] >> shared.task[0].wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task[0].samplesOffs + pos + tid + partSize] >> shared.task[0].wbits : 0;
    const int residualLen = max(0,min(frameSize - pos - shared.task[threadIdx.y].residualOrder, partSize));

    __syncthreads();

    shared.residual[tid] = 0;
    shared.task[threadIdx.y].coefs[threadIdx.x] = threadIdx.x < max_order ? tasks[blockIdx.y * blockDim.y + threadIdx.y].coefs[threadIdx.x] : 0;

    for (int i = blockDim.y * (shared.task[threadIdx.y].type == Verbatim); i < blockDim.y; i++) // += 32
    {
	int ptr = threadIdx.x + (i<<5);
	// compute residual
	int sum = 0;
	int c = 0;
	for (c = 0; c < shared.task[threadIdx.y].residualOrder; c++)
	    sum += __mul24(shared.data[ptr + c], shared.task[threadIdx.y].coefs[c]);
	sum = shared.data[ptr + c] - (sum >> shared.task[threadIdx.y].shift);
	shared.residual[tid] += __mul24(ptr < residualLen, min(0x7fffff,(sum << 1) ^ (sum >> 31)));
    }

    // enable this line when using blockDim.x == 64
    //__syncthreads(); if (threadIdx.x < 32) shared.residual[tid] += shared.residual[tid + 32]; __syncthreads();
    shared.residual[tid] += shared.residual[tid + 16];
    shared.residual[tid] += shared.residual[tid + 8];
    shared.residual[tid] += shared.residual[tid + 4];
    shared.residual[tid] += shared.residual[tid + 2];
    shared.residual[tid] += shared.residual[tid + 1];

    // rice parameter search
    shared.residual[tid] = (shared.task[threadIdx.y].type != Constant || shared.residual[threadIdx.y * blockDim.x] != 0) *
	(__mul24(threadIdx.x >= 15, 0x7fffff) + residualLen * (threadIdx.x + 1) + ((shared.residual[threadIdx.y * blockDim.x] - (residualLen >> 1)) >> threadIdx.x));
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 8]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 4]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 2]);
    shared.residual[tid] = min(shared.residual[tid], shared.residual[tid + 1]);
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid];
}

extern "C" __global__ void cudaChooseBestMethod(
    encodeResidualTaskStruct *tasks,
    int *residual,
    int partCount, // <= blockDim.y (256)
    int taskCount
    )
{
    __shared__ struct {
	volatile int index[128];
	volatile int length[256];
	volatile int partLen[256];
	volatile encodeResidualTaskStruct task[8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    
    shared.length[tid] = 0x7fffffff;
    for (int task = 0; task < taskCount; task += blockDim.y)
	if (task + threadIdx.y < taskCount)
	{
	    // fetch task data
	    ((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(tasks + task + threadIdx.y + taskCount * blockIdx.y))[threadIdx.x];

	    int sum = 0;
	    for (int pos = 0; pos < partCount; pos += blockDim.x)
		sum += (pos + threadIdx.x < partCount ? residual[pos + threadIdx.x + 64 * (task + threadIdx.y + taskCount * blockIdx.y)] : 0);
	    shared.partLen[tid] = sum;

	    // length sum: reduction in shared mem
	    shared.partLen[tid] += shared.partLen[tid + 16];
	    shared.partLen[tid] += shared.partLen[tid + 8];
	    shared.partLen[tid] += shared.partLen[tid + 4];
	    shared.partLen[tid] += shared.partLen[tid + 2];
	    shared.partLen[tid] += shared.partLen[tid + 1];
	    // return sum
	    if (threadIdx.x == 0)
	    {
		int obits = shared.task[threadIdx.y].obits - shared.task[threadIdx.y].wbits;
		shared.length[task + threadIdx.y] =
		    min(obits * shared.task[threadIdx.y].blocksize,
			shared.task[threadIdx.y].type == Fixed ? shared.task[threadIdx.y].residualOrder * obits + 6 + (4 * partCount/2) + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == LPC ? shared.task[threadIdx.y].residualOrder * obits + 4 + 5 + shared.task[threadIdx.y].residualOrder * shared.task[threadIdx.y].cbits + 6 + (4 * partCount/2)/* << porder */ + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].type == Constant ? obits * (1 + shared.task[threadIdx.y].blocksize * (shared.partLen[threadIdx.y * 32] != 0)) : 
			obits * shared.task[threadIdx.y].blocksize);
	    }
	}
    //shared.index[threadIdx.x] = threadIdx.x;
    //shared.length[threadIdx.x] = (threadIdx.x < taskCount) ? tasks[threadIdx.x + taskCount * blockIdx.y].size : 0x7fffffff;

    __syncthreads();

    if (tid < taskCount)
	tasks[tid + taskCount * blockIdx.y].size = shared.length[tid];

    __syncthreads();
    int l1 = shared.length[tid];
    if (tid < 128)
    {
	int l2 = shared.length[tid + 128];
	shared.index[tid] = tid + ((l2 < l1) << 7);
	shared.length[tid] = l1 = min(l1, l2);
    }
    __syncthreads();
    if (tid < 64)
    {
	int l2 = shared.length[tid + 64];
	shared.index[tid] = shared.index[tid + ((l2 < l1) << 6)];
	shared.length[tid] = l1 = min(l1, l2);
    }
    __syncthreads();
    if (tid < 32)
    {
#pragma unroll 5
	for (int sh = 5; sh > 0; sh --)
	{
	    int l2 = shared.length[tid + (1 << sh)];
	    shared.index[tid] = shared.index[tid + ((l2 < l1) << sh)];
	    shared.length[tid] = l1 = min(l1, l2);
	}
	if (tid == 0)
	    tasks[taskCount * blockIdx.y].best_index = taskCount * blockIdx.y + shared.index[shared.length[1] < shared.length[0]];
    }
}

extern "C" __global__ void cudaCopyBestMethod(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index;
    } shared;
    if (threadIdx.x == 0)
	shared.best_index = tasks[count * blockIdx.y].best_index;
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.best_index))[threadIdx.x];
}

extern "C" __global__ void cudaCopyBestMethodStereo(
    encodeResidualTaskStruct *tasks_out,
    encodeResidualTaskStruct *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index[4];
	int best_size[4];
	int lr_index[2];
    } shared;
    if (threadIdx.x < 4)
	shared.best_index[threadIdx.x] = tasks[count * (blockIdx.y * 4 + threadIdx.x)].best_index;
    if (threadIdx.x < 4)
	shared.best_size[threadIdx.x] = tasks[shared.best_index[threadIdx.x]].size;
    __syncthreads();
    if (threadIdx.x == 0)
    {
	int bitsBest = 0x7fffffff;
	if (bitsBest > shared.best_size[2] + shared.best_size[3]) // MidSide
	{
	    bitsBest = shared.best_size[2] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[2];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[3] + shared.best_size[1]) // RightSide
	{
	    bitsBest = shared.best_size[3] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[3];
	    shared.lr_index[1] = shared.best_index[1];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[3]) // LeftSide
	{
	    bitsBest = shared.best_size[0] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[1]) // LeftRight
	{
	    bitsBest = shared.best_size[0] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[1];
	}
    }
    __syncthreads();
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.lr_index[0]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y].residualOffs = tasks[shared.best_index[0]].residualOffs;
    if (threadIdx.x < sizeof(encodeResidualTaskStruct)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y + 1))[threadIdx.x] = ((int*)(tasks + shared.lr_index[1]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y + 1].residualOffs = tasks[shared.best_index[1]].residualOffs;
}

extern "C" __global__ void cudaEncodeResidual(
    int*output,
    int*samples,
    encodeResidualTaskStruct *tasks
    )
{
    __shared__ struct {
	int data[256 + 32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x;
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    __syncthreads();
    const int partSize = blockDim.x;
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(shared.task.blocksize - pos, partSize + shared.task.residualOrder);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task.samplesOffs + pos + tid] >> shared.task.wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task.samplesOffs + pos + tid + partSize] >> shared.task.wbits : 0;
    const int residualLen = max(0,min(shared.task.blocksize - pos - shared.task.residualOrder, partSize));

    __syncthreads();    
    // compute residual
    int sum = 0;
    for (int c = 0; c < shared.task.residualOrder; c++)
	sum += __mul24(shared.data[tid + c], shared.task.coefs[c]);
    __syncthreads();
    shared.data[tid + shared.task.residualOrder] -= (sum >> shared.task.shift);
    __syncthreads();
    if (tid >= shared.task.residualOrder && tid < residualLen + shared.task.residualOrder)
	output[shared.task.residualOffs + pos + tid] = shared.data[tid];
    if (tid + 256 < residualLen + shared.task.residualOrder)
	output[shared.task.residualOffs + pos + tid + 256] = shared.data[tid + 256];
}

extern "C" __global__ void cudaCalcPartition(
    int* partition_lengths,
    int* residual,
    int* samples,
    encodeResidualTaskStruct *tasks,
    int max_porder, // <= 8
    int psize, // == (shared.task.blocksize >> max_porder), < 256
    int parts_per_block // == 256 / psize, > 0, <= 16
    )
{
    __shared__ struct {
	int data[256+32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int parts = min(parts_per_block, (1 << max_porder) - blockIdx.x * parts_per_block);
    const int offs = blockIdx.x * psize * parts_per_block + tid;

    // fetch samples
    if (tid < 32) shared.data[tid] = min(offs, tid + shared.task.residualOrder) >= 32 ? samples[shared.task.samplesOffs + offs - 32] >> shared.task.wbits : 0;
    shared.data[32 + tid] = tid < parts * psize ? samples[shared.task.samplesOffs + offs] >> shared.task.wbits : 0;
    __syncthreads();

    // compute residual
    int s = 0;
    for (int c = -shared.task.residualOrder; c < 0; c++)
	s += __mul24(shared.data[32 + tid + c], shared.task.coefs[shared.task.residualOrder + c]);
    s = shared.data[32 + tid] - (s >> shared.task.shift);

    if (offs >= shared.task.residualOrder && tid < parts * psize)
	residual[shared.task.residualOffs + offs] = s;
    else
	s = 0;

    // convert to unsigned
    s = min(0xfffff, (s << 1) ^ (s >> 31));

    //__syncthreads();
    //shared.data[tid] = s;
    //__syncthreads();

    //shared.data[tid] = (shared.data[tid] & (0x0000ffff << (tid & 16))) | (((shared.data[tid ^ 16] & (0x0000ffff << (tid & 16))) << (~tid & 16)) >> (tid & 16));
    //shared.data[tid] = (shared.data[tid] & (0x00ff00ff << (tid & 8))) | (((shared.data[tid ^ 8] & (0x00ff00ff << (tid & 8))) << (~tid & 8)) >> (tid & 8));
    //shared.data[tid] = (shared.data[tid] & (0x0f0f0f0f << (tid & 4))) | (((shared.data[tid ^ 4] & (0x0f0f0f0f << (tid & 4))) << (~tid & 4)) >> (tid & 4));
    //shared.data[tid] = (shared.data[tid] & (0x33333333 << (tid & 2))) | (((shared.data[tid ^ 2] & (0x33333333 << (tid & 2))) << (~tid & 2)) >> (tid & 2));
    //shared.data[tid] = (shared.data[tid] & (0x55555555 << (tid & 1))) | (((shared.data[tid ^ 1] & (0x55555555 << (tid & 1))) << (~tid & 1)) >> (tid & 1));
    //shared.data[tid] = __popc(shared.data[tid]);

    __syncthreads();
    shared.data[tid + (tid / psize)] = s;
    //shared.data[tid] = s;
    __syncthreads();

    s = (psize - shared.task.residualOrder * (threadIdx.x + blockIdx.x == 0)) * (threadIdx.y + 1);
    int dpos = __mul24(threadIdx.x, psize + 1);
    //int dpos = __mul24(threadIdx.x, psize);
    // calc number of unary bits for part threadIdx.x with rice paramater threadIdx.y
#pragma unroll 0
    for (int i = 0; i < psize; i++)
	s += shared.data[dpos + i] >> threadIdx.y;

    // output length
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x < parts)
	partition_lengths[pos + blockIdx.x * parts_per_block + threadIdx.x] = s;
}

extern "C" __global__ void cudaCalcPartition16(
    int* partition_lengths,
    int* residual,
    int* samples,
    encodeResidualTaskStruct *tasks,
    int max_porder, // <= 8
    int psize, // == 16
    int parts_per_block // == 16
    )
{
    __shared__ struct {
	int data[256+32];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int offs = (blockIdx.x << 8) + tid;

    // fetch samples
    if (tid < 32) shared.data[tid] = min(offs, tid + shared.task.residualOrder) >= 32 ? samples[shared.task.samplesOffs + offs - 32] >> shared.task.wbits : 0;
    shared.data[32 + tid] = samples[shared.task.samplesOffs + offs] >> shared.task.wbits;
    __syncthreads();

    // compute residual
    int s = 0;
    for (int c = -shared.task.residualOrder; c < 0; c++)
	s += __mul24(shared.data[32 + tid + c], shared.task.coefs[shared.task.residualOrder + c]);
    s = shared.data[32 + tid] - (s >> shared.task.shift);

    if (offs >= shared.task.residualOrder)
	residual[shared.task.residualOffs + offs] = s;
    else
	s = 0;

    // convert to unsigned
    s = min(0xfffff, (s << 1) ^ (s >> 31));
    __syncthreads();
    shared.data[tid + threadIdx.y] = s;
    __syncthreads();

    // calc number of unary bits for part threadIdx.x with rice paramater threadIdx.y
    int dpos = __mul24(threadIdx.x, 17);
    s =
    (shared.data[dpos + 0] >> threadIdx.y) + (shared.data[dpos + 1] >> threadIdx.y) + 
    (shared.data[dpos + 2] >> threadIdx.y) + (shared.data[dpos + 3] >> threadIdx.y) + 
    (shared.data[dpos + 4] >> threadIdx.y) + (shared.data[dpos + 5] >> threadIdx.y) + 
    (shared.data[dpos + 6] >> threadIdx.y) + (shared.data[dpos + 7] >> threadIdx.y) + 
    (shared.data[dpos + 8] >> threadIdx.y) + (shared.data[dpos + 9] >> threadIdx.y) + 
    (shared.data[dpos + 10] >> threadIdx.y) + (shared.data[dpos + 11] >> threadIdx.y) + 
    (shared.data[dpos + 12] >> threadIdx.y) + (shared.data[dpos + 13] >> threadIdx.y) + 
    (shared.data[dpos + 14] >> threadIdx.y) + (shared.data[dpos + 15] >> threadIdx.y);

    // output length
    const int pos = ((15 * blockIdx.y + threadIdx.y) << (max_porder + 1)) + (blockIdx.x << 4) + threadIdx.x;
    if (threadIdx.y <= 14)
	partition_lengths[pos] = s + (16 - shared.task.residualOrder * (threadIdx.x + blockIdx.x == 0)) * (threadIdx.y + 1);
}

extern "C" __global__ void cudaCalcLargePartition(
    int* partition_lengths,
    int* residual,
    int* samples,
    encodeResidualTaskStruct *tasks,
    int max_porder, // <= 8
    int psize, // == >= 128
    int parts_per_block // == 1
    )
{
    __shared__ struct {
	int data[256];
	volatile int length[256];
	encodeResidualTaskStruct task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    int sum = 0;
    for (int pos = 0; pos < psize; pos += 256)
    {
	// fetch residual
	int offs = blockIdx.x * psize + pos + tid;
	int s = (offs >= shared.task.residualOrder && pos + tid < psize) ? residual[shared.task.residualOffs + offs] : 0;
	// convert to unsigned
	shared.data[tid] = min(0xfffff, (s << 1) ^ (s >> 31));
	__syncthreads();

	// calc number of unary bits for each residual sample with each rice paramater
#pragma unroll 0
	for (int i = threadIdx.x; i < min(psize,256); i += 16)
	    // for sample (i + threadIdx.x) with this rice paramater (threadIdx.y)
	    sum += shared.data[i] >> threadIdx.y;
	__syncthreads();
    }
    shared.length[tid] = min(0xfffff,sum);
    SUM16(shared.length,tid,+=);

    // output length
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x == 0)
	partition_lengths[pos + blockIdx.x] = min(0xfffff,shared.length[tid]) + (psize - shared.task.residualOrder * (blockIdx.x == 0)) * (threadIdx.y + 1);
}

// Sums partition lengths for a certain k == blockIdx.x
// Requires 128 threads
extern "C" __global__ void cudaSumPartition(
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	volatile int data[512+32]; // max_porder <= 8, data length <= 1 << 9.
    } shared;

    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (blockIdx.x << (max_porder + 1));

    // fetch partition lengths
    shared.data[threadIdx.x] = threadIdx.x < (1 << max_porder) ? partition_lengths[pos + threadIdx.x] : 0;
    shared.data[blockDim.x + threadIdx.x] = blockDim.x + threadIdx.x < (1 << max_porder) ? partition_lengths[pos + blockDim.x + threadIdx.x] : 0;
    __syncthreads();

    int in_pos = (threadIdx.x << 1);
    int out_pos = (1 << max_porder) + threadIdx.x;
    int bs;
    for (bs = 1 << (max_porder - 1); bs > 32; bs >>= 1)
    {
	if (threadIdx.x < bs) shared.data[out_pos] = shared.data[in_pos] + shared.data[in_pos + 1];
	in_pos += bs << 1;
	out_pos += bs;
	__syncthreads();
    }
    if (threadIdx.x < 32)
    for (; bs > 0; bs >>= 1)
    {
	shared.data[out_pos] = shared.data[in_pos] + shared.data[in_pos + 1];
	in_pos += bs << 1;
	out_pos += bs;
    }
    __syncthreads();
    if (threadIdx.x < (1 << max_porder))
	partition_lengths[pos + (1 << max_porder) + threadIdx.x] = shared.data[(1 << max_porder) + threadIdx.x];
    if (blockDim.x + threadIdx.x < (1 << max_porder))
	partition_lengths[pos + (1 << max_porder) + blockDim.x + threadIdx.x] = shared.data[(1 << max_porder) + blockDim.x + threadIdx.x];
}

// Finds optimal rice parameter for up to 16 partitions at a time.
// Requires 16x16 threads
extern "C" __global__ void cudaFindRiceParameter(
    int* rice_parameters,
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	volatile int length[256];
	volatile int index[256];
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 5);
    const int parts = min(32, 2 << max_porder);
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));

    // read length for 32 partitions
    int l1 = (threadIdx.x < parts) ? partition_lengths[pos + blockIdx.x * 32 + threadIdx.x] : 0xffffff;
    int l2 = (threadIdx.y + 8 <= 14 && threadIdx.x < parts) ? partition_lengths[pos + (8 << (max_porder + 1)) + blockIdx.x * 32 + threadIdx.x] : 0xffffff;
    // find best rice parameter
    shared.index[tid] = threadIdx.y + ((l2 < l1) << 3);
    shared.length[tid] = l1 = min(l1, l2);
    __syncthreads();
#pragma unroll 3
    for (int sh = 7; sh >= 5; sh --)
    {
	if (tid < (1 << sh))
	{
	    l2 = shared.length[tid + (1 << sh)];
	    shared.index[tid] = shared.index[tid + ((l2 < l1) << sh)];
	    shared.length[tid] = l1 = min(l1, l2);
	}    
	__syncthreads();
    }
    if (tid < parts)
    {
	// output rice parameter
	rice_parameters[(blockIdx.y << (max_porder + 2)) + blockIdx.x * parts + tid] = shared.index[tid];
	// output length
	rice_parameters[(blockIdx.y << (max_porder + 2)) + (1 << (max_porder + 1)) + blockIdx.x * parts + tid] = shared.length[tid];
    }
}

extern "C" __global__ void cudaFindPartitionOrder(
    int* best_rice_parameters,
    encodeResidualTaskStruct *tasks,
    int* rice_parameters,
    int max_porder
    )
{
    __shared__ struct {
	int data[512];
	volatile int tmp[256];
	int length[32];
	int index[32];
	encodeResidualTaskStruct task;
    } shared;
    const int pos = (blockIdx.y << (max_porder + 2)) + (2 << max_porder);
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    // fetch partition lengths
    shared.data[threadIdx.x] = threadIdx.x < (2 << max_porder) ? rice_parameters[pos + threadIdx.x] : 0;
    shared.data[threadIdx.x + 256] = threadIdx.x + 256 < (2 << max_porder) ? rice_parameters[pos + 256 + threadIdx.x] : 0;
    __syncthreads();

    for (int porder = max_porder; porder >= 0; porder--)
    {
	shared.tmp[threadIdx.x] = (threadIdx.x < (1 << porder)) * shared.data[(2 << max_porder) - (2 << porder) + threadIdx.x];
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x, +=);
	if (threadIdx.x == 0)
	    shared.length[porder] = shared.tmp[0] + (4 << porder);
	__syncthreads();
    }

    if (threadIdx.x < 32)
    {
	shared.index[threadIdx.x] = threadIdx.x;
	if (threadIdx.x > max_porder)
	    shared.length[threadIdx.x] = 0xfffffff;
	int l1 = shared.length[threadIdx.x];
    #pragma unroll 4
	for (int sh = 3; sh >= 0; sh --)
	{
	    int l2 = shared.length[threadIdx.x + (1 << sh)];
	    shared.index[threadIdx.x] = shared.index[threadIdx.x + ((l2 < l1) << sh)];
	    shared.length[threadIdx.x] = l1 = min(l1, l2);
	}
	if (threadIdx.x == 0)
	    tasks[blockIdx.y].porder = shared.index[0];
	if (threadIdx.x == 0)
	{
	    int obits = shared.task.obits - shared.task.wbits;	    
	    tasks[blockIdx.y].size =
		shared.task.type == Fixed ? shared.task.residualOrder * obits + 6 + l1 :
		shared.task.type == LPC ? shared.task.residualOrder * obits + 6 + l1 + 4 + 5 + shared.task.residualOrder * shared.task.cbits :
		shared.task.type == Constant ? obits : obits * shared.task.blocksize;
	}
    }
    __syncthreads();
    int porder = shared.index[0];
    //shared.data[threadIdx.x] = threadIdx.x < (1 << porder) ? rice_parameters[pos - (2 << porder) + threadIdx.x] : 0;
    if (threadIdx.x < (1 << porder))
	best_rice_parameters[(blockIdx.y << max_porder) + threadIdx.x] = rice_parameters[pos - (2 << porder) + threadIdx.x];
    // FIXME: should be bytes?
}

#endif
